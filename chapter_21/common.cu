#include <stdio.h>
#include <hip/hip_runtime.h>

#include "common.cuh"


void gpu_assert(hipError_t code, int line) {
   if (code != hipSuccess) {
      printf("GPUassert: %u %s; line %d\n", code, hipGetErrorString(code), line);
      exit(code);
   }
}


/**
 * Note that this function does not halt the execution.
 */
__device__ void gpu_assert_d(hipError_t code, int line) {
   if (code != hipSuccess) {
      printf("GPUassert: %u %s; line %d\n", code, hipGetErrorString(code), line);
      return;
   }
}