#include <stdio.h>
#include <hip/hip_runtime.h>

#include "common.cuh"


void gpu_assert(hipError_t code, int line) {
   if (code != hipSuccess) {
      printf("GPUassert: %s; line %d\n", hipGetErrorString(code), line);
      exit(code);
   }
}