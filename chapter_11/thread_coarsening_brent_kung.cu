#include <stdio.h>
#include <hip/hip_runtime.h>

#define SECTION_SIZE 2048
#define SUBSECTION_SIZE 4
#define BLOCK_SIZE SECTION_SIZE / SUBSECTION_SIZE / 2
// Here we assume that input size is the same as section size for simplicity.
#define INPUT_SIZE SECTION_SIZE


/**
 *  Perform Brent-Kung kernel for inclusive (segmented) scan with thread coarsening, corresponds to chapter 11.5.
 *  Assume only 1 block is needed in total.
 */
__global__
void BrentKungScanWithThreadCoarseningKernel(
    float* input_array,
    float* output_array,
    unsigned int size
) {
    __shared__ float shared_array[SECTION_SIZE];
    unsigned int global_start_index = threadIdx.x*SUBSECTION_SIZE;

    // Phase 1.1: transfer data from global to shared memory.
    // Recall that two elements are loaded at the beginning in Brent-Kung algorithm.
    for (unsigned int iter = 0; iter < SUBSECTION_SIZE*2; ++iter) {
        unsigned int index = iter*blockDim.x + threadIdx.x;
        if (index < size)
            shared_array[index] = input_array[index];
        else
            shared_array[index] = 0.0f;
    }
    __syncthreads();

    // Phase 1.2: perform sequential scan in each subsection.
    for (unsigned int index = 1; index < SUBSECTION_SIZE; ++index) {
        if (global_start_index+index < size)
            shared_array[global_start_index+index] += shared_array[global_start_index+index-1];
        if (global_start_index+index + blockDim.x*SUBSECTION_SIZE < size)
            shared_array[global_start_index+index + blockDim.x*SUBSECTION_SIZE] += shared_array[global_start_index+index + blockDim.x*SUBSECTION_SIZE-1];


    }
    __syncthreads();

    // Phase 2: perform scan operation on the last element in each subsection.
    for (unsigned int stride = 1; stride <= blockDim.x; stride *= 2) {
        __syncthreads();
        unsigned int subsection_index = (threadIdx.x + 1)*2*stride - 1;
        if (subsection_index < blockDim.x*2)
            shared_array[(subsection_index+1)*SUBSECTION_SIZE-1] += shared_array[(subsection_index-stride+1)*SUBSECTION_SIZE-1];
    }

    for (int stride = SECTION_SIZE/(4*SUBSECTION_SIZE); stride > 0; stride /= 2) {
        __syncthreads();
        unsigned int subsection_index = (threadIdx.x + 1)*stride*2 - 1;
        if (subsection_index + stride < SECTION_SIZE/SUBSECTION_SIZE)
            shared_array[(subsection_index+stride+1)*SUBSECTION_SIZE-1] += shared_array[(subsection_index+1)*SUBSECTION_SIZE-1];
    }

    // Phase 3: add the last element of its predecessor's section to its element except for the last element.
    for (unsigned int index = 0; index < SUBSECTION_SIZE - 1; ++index) {
        if (threadIdx.x > 0)
            shared_array[global_start_index+index] += shared_array[global_start_index-1];
        if (global_start_index+index + blockDim.x*SUBSECTION_SIZE < size)
            shared_array[global_start_index+index + blockDim.x*SUBSECTION_SIZE] += shared_array[global_start_index + blockDim.x*SUBSECTION_SIZE-1];
        __syncthreads();
    }

    // Recall that two elements are stored at the end in Brent-Kung algorithm.
    for (unsigned int iter = 0; iter < SUBSECTION_SIZE*2; ++iter) {
        unsigned int index = iter*blockDim.x + threadIdx.x;
        if (index < size)
            output_array[index] = shared_array[index];
    }
}

void runParallelInclusiveScan(
    float* input_array_h,
    float* output_array_h,
    unsigned int size
) {
    // Get size in bytes.
    size_t size_input = size * sizeof(float);
    size_t size_output = size * sizeof(float);

    // Load and copy input_array and filter to device memory.
    float * input_array_d, * output_array_d;
    
    hipMalloc((void***)&input_array_d, size_input);
    hipMemcpy(input_array_d, input_array_h, size_input, hipMemcpyHostToDevice);

    hipMalloc((void***)&output_array_d, size_output);

    // Invoke kernel.
    dim3 dimBlock(BLOCK_SIZE);
    dim3 dimGrid(1);
    BrentKungScanWithThreadCoarseningKernel<<<dimGrid, dimBlock>>>(input_array_d, output_array_d, size);

    // Copy the output matrix from the device memory.
    hipMemcpy(output_array_h, output_array_d, size_output, hipMemcpyDeviceToHost);

    // Free device vectors.
    hipFree(input_array_d);
    hipFree(output_array_d);
}


int main() {
    unsigned int input_size = INPUT_SIZE;
    float input_array[input_size];
    float output_array[input_size];

    for (int i = 1; i < input_size + 1; ++i) 
        input_array[i-1] = i % 31;

    runParallelInclusiveScan(
        input_array,
        output_array,
        input_size
    );

    // Print the last 3 values of the output array.
    printf("%.0f %.0f %.0f \n", output_array[input_size - 3], output_array[input_size - 2], output_array[input_size - 1]);

    return 0;
}