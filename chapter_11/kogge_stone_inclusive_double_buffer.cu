#include <stdio.h>
#include <hip/hip_runtime.h>

#define SECTION_SIZE 32
#define BLOCK_SIZE SECTION_SIZE
// For now, assume that the input size is the same as the block size.
#define INPUT_SIZE SECTION_SIZE


/**
 *  Perform Kogge-Stone kernel for inclusive (segmented) scan, corresponds to Fig. 11.3 with double-buffering implemented.
 */
__global__
void KoggeStoneScanDoubleBufferKernel(
    float* input_array,
    float* output_array,
    unsigned int size
) {
    __shared__ float buffer_1_array[SECTION_SIZE];
    __shared__ float buffer_2_array[SECTION_SIZE];
    unsigned int global_index = blockIdx.x * blockDim.x + threadIdx.x;

    if (global_index < size)
        buffer_1_array[threadIdx.x] = input_array[global_index];
    else
        buffer_1_array[threadIdx.x] = 0.0f;

    // Store the iteration index to determine which buffer should be read or written.
    unsigned int iter_index = 0;
    for (unsigned int stride = 1; stride < blockDim.x; stride *= 2) {
        __syncthreads();

        if (threadIdx.x >= stride) {
            if (iter_index % 2 == 0)
                buffer_2_array[threadIdx.x] = buffer_1_array[threadIdx.x] + buffer_1_array[threadIdx.x - stride];
            else
                buffer_1_array[threadIdx.x] = buffer_2_array[threadIdx.x] + buffer_2_array[threadIdx.x - stride];
        }

        ++iter_index;
    }

    if (global_index < size) {
        if (iter_index % 2 == 0)
            output_array[global_index] = buffer_1_array[threadIdx.x];
        else
            output_array[global_index] = buffer_2_array[threadIdx.x];
    }
}

void runParallelInclusiveScan(
    float* input_array_h,
    float* output_array_h,
    unsigned int size
) {
    // Get size in bytes.
    size_t size_input = size * sizeof(float);
    size_t size_output = size * sizeof(float);

    // Load and copy input_array and filter to device memory.
    float * input_array_d, * output_array_d;
    
    hipMalloc((void***)&input_array_d, size_input);
    hipMemcpy(input_array_d, input_array_h, size_input, hipMemcpyHostToDevice);

    hipMalloc((void***)&output_array_d, size_output);

    // Invoke kernel.
    dim3 dimBlock(BLOCK_SIZE);
    dim3 dimGrid(ceil(size / BLOCK_SIZE));
    KoggeStoneScanDoubleBufferKernel<<<dimGrid, dimBlock>>>(input_array_d, output_array_d, size);

    // Copy the output matrix from the device memory.
    hipMemcpy(output_array_h, output_array_d, size_output, hipMemcpyDeviceToHost);

    // Free device vectors.
    hipFree(input_array_d);
    hipFree(output_array_d);
}


int main() {
    unsigned int input_size = INPUT_SIZE;
    float input_array[input_size];
    float output_array[input_size];

    for (int i = 1; i < input_size + 1; ++i) 
        input_array[i-1] = i % 31;

    runParallelInclusiveScan(
        input_array,
        output_array,
        input_size
    );

    // Print the last 3 values of the output array.
    printf("%.0f %.0f %.0f \n", output_array[input_size - 3], output_array[input_size - 2], output_array[input_size - 1]);

    return 0;
}