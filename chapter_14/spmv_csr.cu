/**
 *  Perform sparse matrix-vector multiplication using compressed sparse row (csr) format, corresponds to chapter 14.3.
 *  The matrix-vector multiplication performs Y = AX where A denotes the matrix and X denotes the vector.
 */
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>

#define INPUT_LENGTH 8
#define MATRIX_DIM 4
#define THREADS_NUM_PER_BLOCK 4
#define BLOCK_NUM ceil(INPUT_LENGTH * 1.0 / THREADS_NUM_PER_BLOCK)
#define eps 1e-5


__global__
void SparseMatVecMulCsrKernel(
    unsigned int* csr_matrix_row_pointer_indices,
    unsigned int* csr_matrix_col_indices,
    float* csr_matrix_values,
    unsigned int csr_matrix_nonzeroes_size,
    float* vector_X,
    float* vector_Y
) {
    unsigned int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < MATRIX_DIM) {
        float sum = 0.0f;
        for (unsigned int i = csr_matrix_row_pointer_indices[row]; i < csr_matrix_row_pointer_indices[row + 1]; ++i) {
            unsigned int col = csr_matrix_col_indices[i];
            float value = csr_matrix_values[i];
            sum += vector_X[col] * value;
        }
        vector_Y[row] = sum;
    }
}


void runSparseMatVecMultiplication(
    unsigned int* row_pointer_indices_h,
    unsigned int* col_indices_h,
    float* values_h,
    unsigned int num_nonzeroes,
    float* vector_X_h,
    float* vector_Y_h,
    unsigned int vector_dim
) {
    // Load and copy host variables to device memory.
    float *vector_X_d, *vector_Y_d;
    size_t size_array = vector_dim * sizeof(float);
    hipMalloc((void***)&vector_X_d, size_array);
    hipMemcpy(vector_X_d, vector_X_h, size_array, hipMemcpyHostToDevice);
    hipMalloc((void***)&vector_Y_d, size_array);

    unsigned int *row_pointer_indices_d, *col_indices_d;
    float *values_d;
    // Recall that the row pointer array includes one nonexistent row at the end.
    size_t size_array_pointers = (MATRIX_DIM + 1) * sizeof(unsigned int);
    size_t size_array_indices = num_nonzeroes * sizeof(unsigned int);
    size_t size_array_values = num_nonzeroes * sizeof(float);
    
    hipMalloc((void**)&row_pointer_indices_d, size_array_pointers);
    hipMemcpy(row_pointer_indices_d, row_pointer_indices_h, size_array_pointers, hipMemcpyHostToDevice);

    hipMalloc((void**)&col_indices_d, size_array_indices);
    hipMemcpy(col_indices_d, col_indices_h, size_array_indices, hipMemcpyHostToDevice);

    hipMalloc((void**)&values_d, size_array_values);
    hipMemcpy(values_d, values_h, size_array_values, hipMemcpyHostToDevice);

    // Invoke kernel per iteration.
    dim3 dimBlock(THREADS_NUM_PER_BLOCK);
    dim3 dimGrid(BLOCK_NUM);
    
    SparseMatVecMulCsrKernel<<<dimGrid, dimBlock>>>(
        row_pointer_indices_d, col_indices_d, values_d, num_nonzeroes, vector_X_d, vector_Y_d);

    // Copy the output from the device memory.
    hipMemcpy(vector_Y_h, vector_Y_d, size_array, hipMemcpyDeviceToHost);

    // Free device arrays.
    hipFree(vector_X_d);
    hipFree(vector_Y_d);
    hipFree(row_pointer_indices_d);
    hipFree(col_indices_d);
    hipFree(values_d);
}


int main() {
    // Generate the CSR matrix.
    unsigned int row_pointer_indices[] = {0, 2, 5, 7, 8};
    unsigned int col_indices[] = {0, 1, 0, 2, 3, 1, 2, 3};
    float values[] = {1, 7, 5, 3, 9, 2, 8, 6};

    // Prepare vector X for matrix-vector multiplication.
    float X[] = {1, 2, 3, 4};
    float Y_expected[] = {15, 50, 28, 24};
    float Y_actual[MATRIX_DIM];

    runSparseMatVecMultiplication(
        row_pointer_indices,
        col_indices,
        values,
        INPUT_LENGTH,
        X,
        Y_actual,
        MATRIX_DIM
    );

    // Check if the result is correct.
    bool is_correct = true;
    for (int i = 0; i < MATRIX_DIM; ++i)
        if (fabs(Y_actual[i] - Y_expected[i]) > eps) {
            is_correct = false;
            printf("The actual and expected results differ at index %d; actual = %.0f, expected = %.0f\n", i, Y_actual[i], Y_expected[i]);
            break;
        }
    if (is_correct)
        printf("The actual and expected results are identical!\n");

    return 0;
}