#include <stdio.h>
#include <hip/hip_runtime.h>


/**
 *  Perform sum reduction with less control divergence, corresponds to Fig. 10.9.
 */
__global__
void SumReductionLessControlDivergence(
    float* input_array,
    float* output_value,
    unsigned int input_size
) {
    unsigned int index = threadIdx.x;
    // For handling odd size values.
    unsigned int previous_stride = input_size;
    for (unsigned int stride = blockDim.x; stride >= 1; stride = ceil(stride / 2.0)) {
        if (threadIdx.x + stride < previous_stride) {
            input_array[index] += input_array[index + stride];
        }
        // Since ceil(stride / 2.0) = 1, we need to ensure that the loops eventually ends.
        if (stride == 1)
            break;
        previous_stride = stride;
        __syncthreads();
    }
    if (threadIdx.x == 0) {
        *output_value = input_array[0];
    }
}

void runSumReduction(
    float* input_array_h,
    float* output_value_h,
    unsigned int input_size
) {
    // Get size in bytes.
    size_t size_input = input_size * sizeof(float);
    size_t size_output = sizeof(float);

    // Load and copy input_array and filter to device memory.
    float * input_array_d, * output_value_d;
    
    hipMalloc((void***)&input_array_d, size_input);
    hipMemcpy(input_array_d, input_array_h, size_input, hipMemcpyHostToDevice);

    hipMalloc((void***)&output_value_d, size_output);

    unsigned int block_size = ceil(input_size / 2.0);

    // Invoke kernel.
    dim3 dimBlock(block_size);
    dim3 dimGrid(1);
    SumReductionLessControlDivergence<<<dimGrid, dimBlock>>>(input_array_d, output_value_d, input_size);

    // Copy the output matrix from the device memory.
    hipMemcpy(output_value_h, output_value_d, size_output, hipMemcpyDeviceToHost);

    // Free device vectors.
    hipFree(input_array_d);
    hipFree(output_value_d);
}


int main() {
    unsigned int input_size = 20;
    float input_array[input_size];
    float output_value;

    for (int i = 1; i < input_size + 1; ++i) 
        input_array[i-1] = i;

    runSumReduction(
        input_array,
        &output_value,
        input_size
    );

    printf("%.0f\n", output_value);

    return 0;
}