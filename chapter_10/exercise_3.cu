#include <stdio.h>
#include <hip/hip_runtime.h>


/**
 *  Perform sum reduction similar to Fig. 10.9 using the access pattern from exercise 3.
 */
__global__
void SumReductionExercise3(
    float* input_array,
    float* output_value,
    unsigned int input_size
) {
    unsigned int index = 2 * blockDim.x - threadIdx.x - 1;
    for (unsigned int stride = blockDim.x; stride >= 1; stride/=2) {
        if (threadIdx.x < stride) {
            input_array[index] += input_array[index - stride];
        }
        __syncthreads();
    }
    if (threadIdx.x == 0) {
        *output_value = input_array[2 * blockDim.x - 1];
    }
}

void runSumReduction(
    float* input_array_h,
    float* output_value_h,
    unsigned int input_size
) {
    // Get size in bytes.
    size_t size_input = input_size * sizeof(float);
    size_t size_output = sizeof(float);

    // Load and copy input_array and filter to device memory.
    float * input_array_d, * output_value_d;
    
    hipMalloc((void***)&input_array_d, size_input);
    hipMemcpy(input_array_d, input_array_h, size_input, hipMemcpyHostToDevice);

    hipMalloc((void***)&output_value_d, size_output);

    unsigned int block_size = input_size / 2;

    // Invoke kernel.
    dim3 dimBlock(block_size);
    dim3 dimGrid(1);
    SumReductionExercise3<<<dimGrid, dimBlock>>>(input_array_d, output_value_d, input_size);

    // Copy the output matrix from the device memory.
    hipMemcpy(output_value_h, output_value_d, size_output, hipMemcpyDeviceToHost);

    // Free device vectors.
    hipFree(input_array_d);
    hipFree(output_value_d);
}


int main() {
    unsigned int input_size = 1024;
    float input_array[input_size];
    float output_value;

    for (int i = 1; i < input_size + 1; ++i) 
        input_array[i-1] = i;

    runSumReduction(
        input_array,
        &output_value,
        input_size
    );

    printf("%.0f\n", output_value);

    return 0;
}