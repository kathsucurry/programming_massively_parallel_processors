#include <stdio.h>
#include <hip/hip_runtime.h>


#define ORDER 1
// The number of points in the three-dimensional stencil.
#define NUM_POINT 1 + ORDER*2*3
#define BLOCK_SIZE 4
#define IN_TILE_DIM BLOCK_SIZE
#define OUT_TILE_DIM ((IN_TILE_DIM) - 2*(ORDER))


/**
 *  Perform stencil sweep kernel with shared memory tiling, corresponds to Fig. 8.8.
 */
__global__
void StencilKernel(
    float* input_array,
    float* output_array,
    float* coefficients,
    unsigned int size
) {
    int out_z_index = blockIdx.z * OUT_TILE_DIM + threadIdx.z - ORDER;
    int out_y_index = blockIdx.y * OUT_TILE_DIM + threadIdx.y - ORDER;
    int out_x_index = blockIdx.x * OUT_TILE_DIM + threadIdx.x - ORDER;
    int out_index = out_z_index * size * size + out_y_index * size + out_x_index;

    __shared__ float input_shared[IN_TILE_DIM][IN_TILE_DIM][IN_TILE_DIM];
    if (out_z_index >= 0 && out_z_index < size
        && out_y_index >= 0 && out_y_index < size
        && out_x_index >= 0 && out_x_index < size)
        input_shared[threadIdx.z][threadIdx.y][threadIdx.x] = input_array[out_index];

    __syncthreads();

    // Here, we simplify the boundary condition which will not be updated from iteration
    // to the next. Hence, only calculate the inner output grid points.
    if (out_z_index >= 1 && out_z_index < size - 1 
        && out_y_index >= 1 && out_y_index < size - 1
        && out_x_index >= 1 && out_x_index < size - 1)
        if (threadIdx.z >= 1 && threadIdx.z < IN_TILE_DIM - 1
            && threadIdx.y >= 1 && threadIdx.y < IN_TILE_DIM - 1
            && threadIdx.x >= 1 && threadIdx.x < IN_TILE_DIM - 1)
            output_array[out_index] = coefficients[0] * input_shared[threadIdx.z][threadIdx.y][threadIdx.x]
                                    + coefficients[1] * input_shared[threadIdx.z][threadIdx.y][threadIdx.x - 1]
                                    + coefficients[2] * input_shared[threadIdx.z][threadIdx.y][threadIdx.x + 1]
                                    + coefficients[3] * input_shared[threadIdx.z][threadIdx.y - 1][threadIdx.x]
                                    + coefficients[4] * input_shared[threadIdx.z][threadIdx.y + 1][threadIdx.x]
                                    + coefficients[5] * input_shared[threadIdx.z - 1][threadIdx.y][threadIdx.x]
                                    + coefficients[6] * input_shared[threadIdx.z + 1][threadIdx.y][threadIdx.x];
                                    
}


void runStencil(
    float* input_array_h,
    float* output_array_h,
    float* coefficients_h,
    unsigned int size
) {
    // Get size in bytes.
    size_t size_input = size * size * size * sizeof(float);
    size_t size_coefficients = NUM_POINT * sizeof(float);

    // Load and copy input_array and filter to device memory.
    float * input_array_d, * output_array_d, * coefficients_d;
    
    hipMalloc((void***)&input_array_d, size_input);
    hipMemcpy(input_array_d, input_array_h, size_input, hipMemcpyHostToDevice);

    hipMalloc((void***)&coefficients_d, size_coefficients);
    hipMemcpy(coefficients_d, coefficients_h, size_coefficients, hipMemcpyHostToDevice);

    hipMalloc((void***)&output_array_d, size_input);

    // Invoke kernel.
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(
        ceil(size / (OUT_TILE_DIM * 1.0)),
        ceil(size / (OUT_TILE_DIM * 1.0)),
        ceil(size / (OUT_TILE_DIM * 1.0)));
    StencilKernel<<<dimGrid, dimBlock>>>(
        input_array_d,
        output_array_d,
        coefficients_d,
        size
    );

    // Copy the output matrix from the device memory.
    hipMemcpy(output_array_h, output_array_d, size_input, hipMemcpyDeviceToHost);

    // Free device vectors.
    hipFree(input_array_d);
    hipFree(coefficients_d);
    hipFree(output_array_d);
}


int main() {
    // Define input array as a 8 x 8 x 8 array with input (1 .. 512) % 10.
    int size = 8;
    float input_array[size * size * size];

    for (int i = 1; i < size * size * size + 1; ++i) 
        input_array[i-1] = i % 10;


    // Define coefficients.
    float coefficients[] = {1.0f, 2.0f, 3.0f, 4.0f, 5.0f, 6.0f, 7.0f};

    float output_array[size * size * size];

    runStencil(
        input_array,
        output_array,
        coefficients,
        size
    );

    for (int i = 0; i < size; ++i) {
        for (int j = 0; j < size; ++j) {
            for (int k = 0; k < size; ++k) {
                printf("%.0f ", output_array[i * size * size + j * size + k]);
            }
            printf("\n");
        }
        printf("\n\n");
    }

    return 0;
}