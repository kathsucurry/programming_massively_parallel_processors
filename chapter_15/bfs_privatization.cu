/**
 *  Perform BFS with privatization of frontiers, corresponds to chapter 15.6 (Fig. 15.14).
 */
#include <stdio.h>
#include <limits.h>
#include <hip/hip_runtime.h>

#define VERTEX_LENGTH 9
#define LOCAL_FRONTIER_CAPACITY 2
#define THREADS_NUM_PER_BLOCK 4
#define BLOCK_NUM ceil(VERTEX_LENGTH * 1.0 / THREADS_NUM_PER_BLOCK)


struct CSRGraph {
    unsigned int *src_pointer_indices;
    unsigned int *dst_indices;
};


__global__
void BfsFrontierKernel(
    CSRGraph csr_graph,
    unsigned int *level,
    unsigned int *prev_frontier,
    unsigned int *curr_frontier,
    unsigned int num_prev_frontier,
    unsigned int *num_curr_frontier,
    unsigned int curr_level
) {
    // Initialize privatized frontier.
    __shared__ unsigned int shared_curr_frontier[LOCAL_FRONTIER_CAPACITY];
    __shared__ unsigned int shared_num_curr_frontier;
    if (threadIdx.x == 0)
        shared_num_curr_frontier = 0;
    __syncthreads();

    // Perform BFS.
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < num_prev_frontier) {
        unsigned int vertex = prev_frontier[index];
        for (unsigned int edge = csr_graph.src_pointer_indices[vertex];
                edge < csr_graph.src_pointer_indices[vertex + 1]; ++edge) {
            unsigned int neighbor = csr_graph.dst_indices[edge];
            if (atomicCAS(&level[neighbor], UINT_MAX, curr_level) == UINT_MAX) {
                unsigned int shared_curr_frontier_index = atomicAdd(&shared_num_curr_frontier, 1);
                if (shared_curr_frontier_index < LOCAL_FRONTIER_CAPACITY)
                    shared_curr_frontier[shared_curr_frontier_index] = neighbor;
                else {
                    // Recall that shared_num_curr_frontier is currently > LOCAL_FRONTIER_CAPACITY.
                    shared_num_curr_frontier = LOCAL_FRONTIER_CAPACITY;
                    unsigned int curr_frontier_index = atomicAdd(num_curr_frontier, 1);
                    curr_frontier[curr_frontier_index] = neighbor;
                }
            }
        }
    }
    __syncthreads();

    // Allocate in the global frontier.
    __shared__ unsigned int shared_curr_frontier_start_index;
    if (threadIdx.x == 0)
        shared_curr_frontier_start_index = atomicAdd(num_curr_frontier, shared_num_curr_frontier);
    __syncthreads();

    // Commit to the global frontier.
    for (unsigned int thread_frontier_index = threadIdx.x;
            thread_frontier_index < shared_num_curr_frontier;
            thread_frontier_index += blockDim.x) {
        unsigned int curr_frontier_index = shared_curr_frontier_start_index + thread_frontier_index;
        curr_frontier[curr_frontier_index] = shared_curr_frontier[thread_frontier_index];
    }
}


void runBFS(
    unsigned int *csr_src_pointer_indices_h,
    unsigned int *csr_dst_indices_h,
    unsigned int edge_length,
    unsigned int *level_h,
    unsigned int root
) {
    // Load and copy host variables to device memory.
    unsigned int *csr_src_pointer_indices_d, *csr_dst_indices_d, *level_d;
    // Frontier-related device variables.
    unsigned int *prev_frontier_d, *curr_frontier_d, *num_curr_frontier_d, *num_prev_frontier_d;

    size_t size_src_pointer_indices = (VERTEX_LENGTH + 1) * sizeof(unsigned int);
    size_t size_dst_indices = edge_length * sizeof(unsigned int);
    size_t size_vertex = VERTEX_LENGTH * sizeof(unsigned int);

    hipMalloc((void**)&csr_src_pointer_indices_d, size_src_pointer_indices);
    hipMemcpy(csr_src_pointer_indices_d, csr_src_pointer_indices_h, size_src_pointer_indices, hipMemcpyHostToDevice);

    hipMalloc((void**)&csr_dst_indices_d, size_dst_indices);
    hipMemcpy(csr_dst_indices_d, csr_dst_indices_h, size_dst_indices, hipMemcpyHostToDevice);

    struct CSRGraph csr_graph_d = {
        .src_pointer_indices = csr_src_pointer_indices_d,
        .dst_indices = csr_dst_indices_d
    };

    hipMalloc((void**)&level_d, size_vertex);
    // Initialize with "not visited" value.
    hipMemset(level_d, UINT_MAX, size_vertex);

    hipMalloc((void**)&prev_frontier_d, size_vertex);
    hipMalloc((void**)&curr_frontier_d, size_vertex);
    hipMalloc((void**)&num_curr_frontier_d, sizeof(unsigned int));
    hipMalloc((void**)&num_prev_frontier_d, sizeof(unsigned int));

    // Invoke kernel per iteration.
    dim3 dimBlock(THREADS_NUM_PER_BLOCK);
    dim3 dimGrid(BLOCK_NUM);

    // Assign level[root] to 0 (i.e., visited).
    hipMemset(level_d + root, 0, sizeof(unsigned int));
    unsigned int curr_level = 1;
    
    unsigned int *num_prev_frontier_h = (unsigned int*)malloc(sizeof(unsigned int));
    num_prev_frontier_h[0] = 1;
    hipMemcpy(num_prev_frontier_d, num_prev_frontier_h, sizeof(unsigned int), hipMemcpyHostToDevice);

    unsigned int *prev_frontier_h = (unsigned int*)malloc(VERTEX_LENGTH * sizeof(unsigned int));
    prev_frontier_h[0] = root;    
    hipMemcpy(prev_frontier_d, prev_frontier_h, VERTEX_LENGTH * sizeof(unsigned int), hipMemcpyHostToDevice);

    while (num_prev_frontier_h[0] != 0) {
        hipMemset(num_curr_frontier_d, 0, sizeof(unsigned int));

        BfsFrontierKernel<<<dimGrid, dimBlock>>>(
            csr_graph_d,
            level_d,
            prev_frontier_d,
            curr_frontier_d,
            num_prev_frontier_h[0],
            num_curr_frontier_d,
            curr_level
        );
        hipMemcpy(num_prev_frontier_h, num_curr_frontier_d, sizeof(unsigned int), hipMemcpyDeviceToHost);

        // Switch prev and curr frontiers.
        unsigned int *temp = prev_frontier_d;
        prev_frontier_d = curr_frontier_d;
        curr_frontier_d = temp;

        temp = num_prev_frontier_d;
        num_prev_frontier_d = num_curr_frontier_d;
        num_curr_frontier_d = temp;

        ++curr_level;
    }

    // Copy level from the device memory.
    hipMemcpy(level_h, level_d, size_vertex, hipMemcpyDeviceToHost);

    free(num_prev_frontier_h);
    free(prev_frontier_h);

    // Free device arrays.
    hipFree(csr_src_pointer_indices_d);
    hipFree(csr_dst_indices_d);
    hipFree(level_d);
    hipFree(prev_frontier_d);
    hipFree(curr_frontier_d);
    hipFree(num_prev_frontier_d);
    hipFree(num_curr_frontier_d);
}


int main() {
    // Generate the CSR graph; all values (edge weight) are 1s.
    unsigned int src_pointer_indices[] = {0, 2, 4, 7, 9, 11, 12, 13, 15, 15};
    unsigned int dst_indices[] = {1, 2, 3, 4, 5, 6, 7, 4, 8, 5, 8, 6, 8, 0, 6};
    unsigned int edge_length = 15;

    unsigned int root = 2;
    unsigned int *level = (unsigned int*)malloc(VERTEX_LENGTH * sizeof(unsigned int));
    runBFS(
        src_pointer_indices,
        dst_indices,
        edge_length,
        level,
        root
    );

    for (unsigned int i = 0; i < VERTEX_LENGTH; ++i)
        printf("%u ", level[i]);
    printf("\n");

    return 0;
}