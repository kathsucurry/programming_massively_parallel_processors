/**
 *  Perform BFS with privatization of frontiers + single-block kernel;
 *  corresponds to chapter 15.7 (Fig. 15.16).
 */
#include <stdio.h>
#include <limits.h>
#include <hip/hip_runtime.h>

#define VERTEX_LENGTH 15
#define LOCAL_FRONTIER_CAPACITY 3
#define THREADS_NUM_PER_BLOCK 4
#define BLOCK_NUM ceil(VERTEX_LENGTH * 1.0 / THREADS_NUM_PER_BLOCK)


struct CSRGraph {
    unsigned int *src_pointer_indices;
    unsigned int *dst_indices;
};


__global__
void BfsFrontierKernel(
    CSRGraph csr_graph,
    unsigned int *level,
    unsigned int *prev_frontier,
    unsigned int *curr_frontier,
    unsigned int num_prev_frontier,
    unsigned int *num_curr_frontier,
    unsigned int curr_level
) {
    // Initialize privatized frontier.
    __shared__ unsigned int shared_curr_frontier[LOCAL_FRONTIER_CAPACITY];
    __shared__ unsigned int shared_num_curr_frontier;
    if (threadIdx.x == 0)
        shared_num_curr_frontier = 0;
    __syncthreads();

    // Perform BFS.
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < num_prev_frontier) {
        unsigned int vertex = prev_frontier[index];
        for (unsigned int edge = csr_graph.src_pointer_indices[vertex];
                edge < csr_graph.src_pointer_indices[vertex + 1]; ++edge) {
            unsigned int neighbor = csr_graph.dst_indices[edge];
            if (atomicCAS(&level[neighbor], UINT_MAX, curr_level) == UINT_MAX) {
                unsigned int shared_curr_frontier_index = atomicAdd(&shared_num_curr_frontier, 1);
                if (shared_curr_frontier_index < LOCAL_FRONTIER_CAPACITY)
                    shared_curr_frontier[shared_curr_frontier_index] = neighbor;
                else {
                    // Recall that shared_num_curr_frontier is currently > LOCAL_FRONTIER_CAPACITY.
                    shared_num_curr_frontier = LOCAL_FRONTIER_CAPACITY;
                    unsigned int curr_frontier_index = atomicAdd(num_curr_frontier, 1);
                    curr_frontier[curr_frontier_index] = neighbor;
                }
            }
        }
    }
    __syncthreads();

    // Allocate in the global frontier.
    __shared__ unsigned int shared_curr_frontier_start_index;
    if (threadIdx.x == 0)
        shared_curr_frontier_start_index = atomicAdd(num_curr_frontier, shared_num_curr_frontier);
    __syncthreads();

    // Commit to the global frontier.
    for (unsigned int thread_frontier_index = threadIdx.x;
            thread_frontier_index < shared_num_curr_frontier;
            thread_frontier_index += blockDim.x) {
        unsigned int curr_frontier_index = shared_curr_frontier_start_index + thread_frontier_index;
        curr_frontier[curr_frontier_index] = shared_curr_frontier[thread_frontier_index];
    }
}


__global__
void BfsSingleBlockKernel(
    CSRGraph csr_graph,
    unsigned int *level,
    unsigned int *prev_frontier,
    unsigned int *curr_frontier,
    unsigned int num_prev_frontier,
    unsigned int *num_curr_frontier,
    unsigned int *curr_level
) {
    // Initialize privatized frontier.
    __shared__ unsigned int shared_curr_frontier[LOCAL_FRONTIER_CAPACITY];
    __shared__ unsigned int shared_num_curr_frontier;
    __shared__ unsigned int shared_prev_frontier[LOCAL_FRONTIER_CAPACITY];
    __shared__ unsigned int shared_num_prev_frontier;
    // A shared variable to determine whether it's time to change kernel (if value = 1).
    __shared__ unsigned int shared_change_kernel;

    if (threadIdx.x == 0) {
        shared_change_kernel = 0;
        shared_num_curr_frontier = 0;
        shared_num_prev_frontier = num_prev_frontier;
    }
    __syncthreads();

    // Perform BFS.
    if (threadIdx.x < shared_num_prev_frontier)
        shared_prev_frontier[threadIdx.x] = prev_frontier[threadIdx.x];

    while (true) {
        if (threadIdx.x < shared_num_prev_frontier) {
            unsigned int vertex = shared_prev_frontier[threadIdx.x];
            for (unsigned int edge = csr_graph.src_pointer_indices[vertex];
                    edge < csr_graph.src_pointer_indices[vertex + 1]; ++edge) {
                unsigned int neighbor = csr_graph.dst_indices[edge];
                if (atomicCAS(&level[neighbor], UINT_MAX, curr_level[0]) == UINT_MAX) {
                    unsigned int shared_curr_frontier_index = atomicAdd(&shared_num_curr_frontier, 1);
                    if (shared_curr_frontier_index < LOCAL_FRONTIER_CAPACITY)
                        shared_curr_frontier[shared_curr_frontier_index] = neighbor;
                    else {
                        // Recall that shared_num_curr_frontier is currently > LOCAL_FRONTIER_CAPACITY.
                        shared_num_curr_frontier = LOCAL_FRONTIER_CAPACITY;
                        unsigned int curr_frontier_index = atomicAdd(num_curr_frontier, 1);
                        curr_frontier[curr_frontier_index] = neighbor;
                        // Time to change kernel.
                        shared_change_kernel = 1;
                    }
                }
            }
        }
        __syncthreads();

        if (shared_change_kernel != 0 || shared_num_curr_frontier == 0)
            break;

        if (threadIdx.x == 0) {
            curr_level[0]++;
            shared_num_prev_frontier = shared_num_curr_frontier;
            shared_num_curr_frontier = 0;
        }
        __syncthreads();

        if (threadIdx.x < shared_num_prev_frontier)
            shared_prev_frontier[threadIdx.x] = shared_curr_frontier[threadIdx.x];
        __syncthreads();
    }

    // Allocate in the global frontier.
    __shared__ unsigned int shared_curr_frontier_start_index;
    if (threadIdx.x == 0)
        shared_curr_frontier_start_index = atomicAdd(num_curr_frontier, shared_num_curr_frontier);
    __syncthreads();

    // Commit to the global frontier.
    for (unsigned int thread_frontier_index = threadIdx.x;
            thread_frontier_index < shared_num_curr_frontier;
            thread_frontier_index += blockDim.x) {
        unsigned int curr_frontier_index = shared_curr_frontier_start_index + thread_frontier_index;
        curr_frontier[curr_frontier_index] = shared_curr_frontier[thread_frontier_index];
    }
}


void runBFS(
    unsigned int *csr_src_pointer_indices_h,
    unsigned int *csr_dst_indices_h,
    unsigned int edge_length,
    unsigned int *level_h,
    unsigned int root
) {
    // Load and copy host variables to device memory.
    unsigned int *csr_src_pointer_indices_d, *csr_dst_indices_d, *level_d;
    // Frontier-related device variables.
    unsigned int *prev_frontier_d, *curr_frontier_d, *num_curr_frontier_d, *num_prev_frontier_d;
    // Single-block related device variables.
    unsigned int *curr_level_d;

    size_t size_src_pointer_indices = (VERTEX_LENGTH + 1) * sizeof(unsigned int);
    size_t size_dst_indices = edge_length * sizeof(unsigned int);
    size_t size_vertex = VERTEX_LENGTH * sizeof(unsigned int);

    hipMalloc((void**)&csr_src_pointer_indices_d, size_src_pointer_indices);
    hipMemcpy(csr_src_pointer_indices_d, csr_src_pointer_indices_h, size_src_pointer_indices, hipMemcpyHostToDevice);

    hipMalloc((void**)&csr_dst_indices_d, size_dst_indices);
    hipMemcpy(csr_dst_indices_d, csr_dst_indices_h, size_dst_indices, hipMemcpyHostToDevice);

    struct CSRGraph csr_graph_d = {
        .src_pointer_indices = csr_src_pointer_indices_d,
        .dst_indices = csr_dst_indices_d
    };

    hipMalloc((void**)&level_d, size_vertex);
    // Initialize with "not visited" value.
    hipMemset(level_d, UINT_MAX, size_vertex);

    hipMalloc((void**)&prev_frontier_d, size_vertex);
    hipMalloc((void**)&curr_frontier_d, size_vertex);
    hipMalloc((void**)&num_curr_frontier_d, sizeof(unsigned int));
    hipMalloc((void**)&num_prev_frontier_d, sizeof(unsigned int));

    // Prepare for the kernel launch:
    // 1) Assign level[root] to 0 (i.e., visited).
    hipMemset(level_d + root, 0, sizeof(unsigned int));
    
    // 2) Update previous frontiers.
    unsigned int *num_prev_frontier_h = (unsigned int*)malloc(sizeof(unsigned int));
    num_prev_frontier_h[0] = 1;
    hipMemcpy(num_prev_frontier_d, num_prev_frontier_h, sizeof(unsigned int), hipMemcpyHostToDevice);

    unsigned int *prev_frontier_h = (unsigned int*)malloc(VERTEX_LENGTH * sizeof(unsigned int));
    prev_frontier_h[0] = root;    
    hipMemcpy(prev_frontier_d, prev_frontier_h, VERTEX_LENGTH * sizeof(unsigned int), hipMemcpyHostToDevice);

    // 3) Set up current level.
    unsigned int *curr_level_h = (unsigned int*)malloc(sizeof(unsigned int));
    curr_level_h[0] = 0;
    hipMalloc((void**)&curr_level_d, sizeof(unsigned int));

    while (num_prev_frontier_h[0] != 0) {
        hipMemset(num_curr_frontier_d, 0, sizeof(unsigned int));
        
        ++curr_level_h[0];

        if (num_prev_frontier_h[0] <= LOCAL_FRONTIER_CAPACITY) {
            hipMemcpy(curr_level_d, curr_level_h, sizeof(unsigned int), hipMemcpyHostToDevice);
            dim3 dimBlock(LOCAL_FRONTIER_CAPACITY);
            dim3 dimGrid(1);
            BfsSingleBlockKernel<<<dimGrid, dimBlock>>>(
                csr_graph_d,
                level_d,
                prev_frontier_d,
                curr_frontier_d,
                num_prev_frontier_h[0],
                num_curr_frontier_d,
                curr_level_d
            );
            hipMemcpy(curr_level_h, curr_level_d, sizeof(unsigned int), hipMemcpyDeviceToHost);
        } else {
            // Invoke kernel per iteration.
            dim3 dimBlock(THREADS_NUM_PER_BLOCK);
            dim3 dimGrid(BLOCK_NUM);
            BfsFrontierKernel<<<dimGrid, dimBlock>>>(
                csr_graph_d,
                level_d,
                prev_frontier_d,
                curr_frontier_d,
                num_prev_frontier_h[0],
                num_curr_frontier_d,
                curr_level_h[0]
            );
        }
        hipMemcpy(num_prev_frontier_h, num_curr_frontier_d, sizeof(unsigned int), hipMemcpyDeviceToHost);

        // Switch prev and curr frontiers.
        unsigned int *temp = prev_frontier_d;
        prev_frontier_d = curr_frontier_d;
        curr_frontier_d = temp;

        temp = num_prev_frontier_d;
        num_prev_frontier_d = num_curr_frontier_d;
        num_curr_frontier_d = temp;
    }

    // Copy level from the device memory.
    hipMemcpy(level_h, level_d, size_vertex, hipMemcpyDeviceToHost);

    free(num_prev_frontier_h);
    free(prev_frontier_h);

    // Free device arrays.
    hipFree(csr_src_pointer_indices_d);
    hipFree(csr_dst_indices_d);
    hipFree(level_d);
    hipFree(prev_frontier_d);
    hipFree(curr_frontier_d);
    hipFree(num_prev_frontier_d);
    hipFree(num_curr_frontier_d);
}


int main() {
    // Generate the CSR graph; all values (edge weight) are 1s.
    unsigned int src_pointer_indices[] = {0, 2, 4, 7, 8, 8, 11, 14, 15, 17, 19, 19, 20};
    unsigned int dst_indices[] = {1, 2, 3, 4, 4, 5, 6, 7, 7, 8, 10, 0, 9, 13, 11, 11, 12, 10, 12, 14};
    unsigned int edge_length = 20;

    unsigned int root = 0;
    unsigned int *level = (unsigned int*)malloc(VERTEX_LENGTH * sizeof(unsigned int));
    runBFS(
        src_pointer_indices,
        dst_indices,
        edge_length,
        level,
        root
    );

    for (unsigned int i = 0; i < VERTEX_LENGTH; ++i)
        printf("%u ", level[i]);
    printf("\n");

    return 0;
}