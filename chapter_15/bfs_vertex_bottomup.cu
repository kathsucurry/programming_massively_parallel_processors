/**
 *  Perform BFS bottom up, corresponds to chapter 15.3 (Fig. 15.8).
 */
#include <stdio.h>
#include <limits.h>
#include <hip/hip_runtime.h>

#define VERTEX_LENGTH 9
#define THREADS_NUM_PER_BLOCK 4
#define BLOCK_NUM ceil(VERTEX_LENGTH * 1.0 / THREADS_NUM_PER_BLOCK)


__global__
void BfsBottomUpKernel(
    unsigned int *dst_pointer_indices,
    unsigned int *src_indices,
    unsigned int *level,
    unsigned int current_level,
    unsigned int *new_vertex_visited
) {
    unsigned int vertex = blockIdx.x * blockDim.x + threadIdx.x;
    if (vertex < VERTEX_LENGTH && level[vertex] == UINT_MAX) {
        for (unsigned int edge = dst_pointer_indices[vertex];
            edge < dst_pointer_indices[vertex + 1]; ++edge) {
                unsigned int neighbor = src_indices[edge];
                if (level[neighbor] == current_level - 1) {
                    level[vertex] = current_level;
                    new_vertex_visited[0] = 1;
                    break;
                }
        }
    }
}


void runBFS(
    unsigned int *csc_dst_pointer_indices_h,
    unsigned int *csc_src_indices_h,
    unsigned int edge_length,
    unsigned int *level_h,
    unsigned int root
) {
    // Load and copy host variables to device memory.
    unsigned int *csc_dst_pointer_indices_d, *csc_src_indices_d, *level_d, *new_vertex_visited_d;

    size_t size_dst_pointer_indices = (VERTEX_LENGTH + 1) * sizeof(unsigned int);
    size_t size_src_indices = edge_length * sizeof(unsigned int);
    size_t size_vertex = VERTEX_LENGTH * sizeof(unsigned int);

    hipMalloc((void**)&csc_dst_pointer_indices_d, size_dst_pointer_indices);
    hipMemcpy(csc_dst_pointer_indices_d, csc_dst_pointer_indices_h, size_dst_pointer_indices, hipMemcpyHostToDevice);

    hipMalloc((void**)&csc_src_indices_d, size_src_indices);
    hipMemcpy(csc_src_indices_d, csc_src_indices_h, size_src_indices, hipMemcpyHostToDevice);

    hipMalloc((void**)&level_d, size_vertex);
    // Initialize with "not visited" value.
    hipMemset(level_d, UINT_MAX, size_vertex);

    // new_vertex_visited is used to determine whether we need to do another iteration.
    hipMalloc((void**)&new_vertex_visited_d, sizeof(unsigned int));

    // Invoke kernel per iteration.
    dim3 dimBlock(THREADS_NUM_PER_BLOCK);
    dim3 dimGrid(BLOCK_NUM);

    unsigned int *new_vertex_visited_h = (unsigned int*)malloc(sizeof(unsigned int));
    // Initialize with 1.
    new_vertex_visited_h[0] = 1;

    // Assign level[root] to 0 (i.e., visited).
    hipMemset(level_d + root, 0, sizeof(unsigned int));
    unsigned int current_level = 1;

    while (new_vertex_visited_h[0] != 0) {
        hipMemset(new_vertex_visited_d, 0, sizeof(unsigned int));

        BfsBottomUpKernel<<<dimGrid, dimBlock>>>(
            csc_dst_pointer_indices_d,
            csc_src_indices_d,
            level_d,
            current_level,
            new_vertex_visited_d
        );

        hipMemcpy(new_vertex_visited_h, new_vertex_visited_d, sizeof(unsigned int), hipMemcpyDeviceToHost);
        ++current_level;
    }

    free(new_vertex_visited_h);

    // Copy level from the device memory.
    hipMemcpy(level_h, level_d, size_vertex, hipMemcpyDeviceToHost);

    // Free device arrays.
    hipFree(csc_dst_pointer_indices_d);
    hipFree(csc_src_indices_d);
    hipFree(level_d);
    hipFree(new_vertex_visited_d);
}


int main() {
    // Generate the CSC graph; all values (edge weight) are 1s.
    unsigned int dst_pointer_indices[] = {0, 1, 2, 3, 4, 6, 8, 11, 12, 15};
    unsigned int src_indices[] = {7, 0, 0, 1, 1, 3, 2, 4, 2, 5, 7, 2, 3, 4, 6};
    unsigned int edge_length = 15;

    unsigned int root = 2;
    unsigned int *level = (unsigned int*)malloc(VERTEX_LENGTH * sizeof(unsigned int));
    runBFS(
        dst_pointer_indices,
        src_indices,
        edge_length,
        level,
        root
    );

    for (unsigned int i = 0; i < VERTEX_LENGTH; ++i)
        printf("%u ", level[i]);
    printf("\n");

    return 0;
}