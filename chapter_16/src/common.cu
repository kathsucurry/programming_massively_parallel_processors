#include <hip/hip_runtime.h>
#include <stdio.h> 


hipError_t cudaMallocCheck(void **dev_ptr, size_t size) {
    hipError_t cuda_malloc_return = hipMalloc(dev_ptr, size);
    if (cuda_malloc_return == hipErrorOutOfMemory) {
        printf("CUDA malloc failed!\n");
        exit(EXIT_FAILURE);
    }
    return cuda_malloc_return;
}


void *mallocCheck(size_t size) {
    void *malloc_return = malloc(size);
    if (malloc_return == NULL) {
        printf("Malloc failed!\n");
        exit(EXIT_FAILURE);
    }
    return malloc_return;
}
