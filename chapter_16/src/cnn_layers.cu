#include "hip/hip_runtime.h"
#include <stdint.h>
#include <stdlib.h>
#include <math.h>
#include <stdio.h>
#include <hip/hip_runtime.h>

#include "cnn_layers.cuh"
#include "kernel_functions.cuh"
#include "common.h"


Tensor *initialize_tensor() {
    Tensor *tensor = (Tensor *)malloc(sizeof(Tensor));
    tensor->num_dim = 0;
    tensor->dim = NULL;
    tensor->values_d = NULL;
    return tensor;
}


void free_tensor(Tensor *tensor) {
    hipFree(tensor->values_d);
    free(tensor->dim);
    free(tensor);
}


float *_uniform_xavier_initialization(uint32_t fan_in, uint32_t fan_out, uint32_t size, uint32_t seed) {
    // Assume gain = 1.
    srand(seed);
    float x = sqrtf(6.0 / (fan_in + fan_out));
    float *array = (float *)malloc(size * sizeof(float));
    for (uint32_t i = 0; i < size; ++i)
        array[i] = x * 2 * (rand() * 1.0 / RAND_MAX) - x; 
    return array;
}


// For simplicity, assume stride is always 1.
Tensor *initialize_conv_layer_weights(
    uint32_t in_channels,
    uint32_t out_channels,
    uint8_t filter_length,
    uint32_t seed
) {
    Tensor *conv_weight = (Tensor *)malloc(sizeof(Tensor));
    // Dimensions = out_channels * in_channels * filter_length * filter_length.
    conv_weight->num_dim = 4;
    uint32_t *dim = (uint32_t *)malloc(conv_weight->num_dim * sizeof(uint32_t));
    dim[0] = out_channels;
    dim[1] = in_channels;
    dim[2] = filter_length;
    dim[3] = filter_length;
    conv_weight->dim = dim;

    uint32_t weight_size = out_channels * in_channels * filter_length * filter_length;
    uint32_t fan_in = in_channels * filter_length * filter_length;
    uint32_t fan_out = out_channels * filter_length * filter_length;

    float *filters = _uniform_xavier_initialization(fan_in, fan_out, weight_size, seed);
    float *filters_d;
    hipMalloc((void**)&filters_d, weight_size * sizeof(float));
    hipMemcpy(filters_d, filters, weight_size * sizeof(float), hipMemcpyHostToDevice);
    conv_weight->values_d = filters_d;
    free(filters);
    
    return conv_weight;
}


Tensor *initialize_linear_layer_weights(uint32_t in_features, uint32_t out_features, uint32_t seed) {
    Tensor *linear_weight = (Tensor *)malloc(sizeof(Tensor));
    linear_weight->num_dim = 2;

    uint32_t *dim = (uint32_t *)malloc(linear_weight->num_dim * sizeof(uint32_t));
    dim[0] = out_features;
    dim[1] = in_features;
    linear_weight->dim = dim;
    uint32_t weight_size = out_features * in_features;

    float *weights = _uniform_xavier_initialization(in_features, out_features, weight_size, seed);
    float *weights_d;
    hipMalloc((void**)&weights_d, weight_size * sizeof(float));
    hipMemcpy(weights_d, weights, weight_size * sizeof(float), hipMemcpyHostToDevice);
    linear_weight->values_d = weights_d;
    free(weights);

    return linear_weight;
}


uint32_t get_tensor_values_size(const uint8_t num_dim, const uint32_t *dim) {
    uint32_t size = 1;
    for (uint8_t i = 0; i < num_dim; ++i)
        size *= dim[i];
    return size;
}


/**
 * Conv2 kernel implementation, following the tiled method in chapter 16.3
 */
void run_conv2d_forward(
    Tensor *output,
    float *X_d,
    Tensor *filters,
    uint32_t num_samples,
    uint32_t in_height,
    uint32_t in_width
) {
    uint32_t filter_length = filters->dim[filters->num_dim - 2];
    uint32_t out_height    = in_height - filter_length + 1;
    uint32_t out_width     = in_width - filter_length + 1;
    uint32_t out_channels  = filters->dim[0];
    uint32_t in_channels   = filters->dim[1];
    uint32_t out_size      = num_samples * out_channels * out_height * out_width;

    float *Y_d;
    hipMalloc((void**)&Y_d, out_size * sizeof(float));

    uint32_t grid_width = ceil(out_width * 1.0 / TILE_WIDTH);
    uint32_t grid_height = ceil(out_height * 1.0 / TILE_WIDTH);
    uint32_t out_tiles_num = grid_width * grid_height;

    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);
    dim3 dimGrid(out_channels, out_tiles_num, num_samples);
    Conv2ForwardKernel<<<dimGrid, dimBlock>>>(
        X_d, Y_d,
        filters->values_d,
        filter_length,
        in_channels,
        grid_height, grid_width,
        in_height, in_width,
        out_height, out_width
    );

    output->num_dim = 4;

    uint32_t *dim = (uint32_t *)malloc(output->num_dim * sizeof(uint32_t));
    dim[0] = num_samples;
    dim[1] = out_channels;
    dim[2] = out_height;
    dim[3] = out_width;

    output->dim = dim;
    output->values_d = Y_d;
}


void run_sigmoid_forward(Tensor *tensor) {
    uint32_t num_samples    = tensor->dim[0];
    uint32_t num_channels   = tensor->dim[1];
    uint32_t feature_height = tensor->dim[2];
    uint32_t feature_width  = tensor->dim[3];
    uint32_t out_size       = num_samples * num_channels * feature_height * feature_width;

    float *Y_d;
    hipMalloc((void**)&Y_d, out_size * sizeof(float));

    uint32_t grid_height = ceil(feature_height * 1.0 / TILE_WIDTH);
    uint32_t grid_width = ceil(feature_width * 1.0 / TILE_WIDTH);
    uint32_t out_tiles_num = grid_width * grid_height;

    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);
    dim3 dimGrid(num_channels, out_tiles_num, num_samples);

    SigmoidForwardKernel<<<dimGrid, dimBlock>>>(
        tensor->values_d, Y_d,
        grid_height, grid_width,
        feature_height, feature_width
    );

    // Update tensor.
    hipFree(tensor->values_d);
    tensor->values_d = Y_d;
}


// Assume stride is always the kernel size.
void run_pooling_forward(Tensor *tensor, uint32_t kernel_length, pooling_type pool_type) {
    uint32_t num_samples    = tensor->dim[0];
    uint32_t num_channels   = tensor->dim[1];
    uint32_t feature_height = tensor->dim[2];
    uint32_t feature_width  = tensor->dim[3];
    uint32_t out_height     = feature_height / kernel_length;
    uint32_t out_width      = feature_width / kernel_length;
    uint32_t out_size       = num_samples * num_channels * out_height * out_width;

    float *Y_d;
    hipMalloc((void**)&Y_d, out_size * sizeof(float));

    uint32_t grid_height = ceil(out_height * 1.0 / TILE_WIDTH);
    uint32_t grid_width = ceil(out_width * 1.0 / TILE_WIDTH);
    uint32_t out_tiles_num = grid_width * grid_height;

    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);
    dim3 dimGrid(num_channels, out_tiles_num, num_samples);

    if (pool_type != MEAN && pool_type != MAX) {
        printf("The inputted pooling type is currently not implemented.");
        free_tensor(tensor);
        hipFree(Y_d);
        return;
    }

    PoolForwardKernel<<<dimGrid, dimBlock>>>(
        tensor->values_d, Y_d,
        pool_type,
        kernel_length,
        grid_height, grid_width,
        feature_height, feature_width,
        out_height, out_width
    );

    // Update tensor.
    tensor->dim[2] = out_height;
    tensor->dim[3] = out_width;
    hipFree(tensor->values_d);
    tensor->values_d = Y_d;
}


void run_flatten_layer(Tensor *tensor) {
    // Make sure to keep the sample dimension.
    uint32_t num_samples = tensor->dim[0];
    uint32_t size = get_tensor_values_size(tensor->num_dim, tensor->dim) / num_samples;
    tensor->num_dim = 2;
    free(tensor->dim);

    uint32_t *dim = (uint32_t *)malloc(tensor->num_dim * sizeof(uint32_t));
    dim[0] = num_samples;
    dim[1] = size;
    tensor->dim = dim;
}


void run_linear_forward(Tensor *X, Tensor *linear_weights) {
    uint32_t in_features  = linear_weights->dim[1];
    uint32_t out_features = linear_weights->dim[0];
    uint32_t num_samples  = X->dim[0];
    uint32_t out_size     = num_samples * out_features;

    float *Y_d;
    hipMalloc((void**)&Y_d, out_size * sizeof(float));

    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);
    dim3 dimGrid(ceil(out_features * 1.0 / (TILE_WIDTH * THREAD_COARSENING_FACTOR)), ceil(num_samples * 1.0 / TILE_WIDTH));
    LinearForwardKernel<<<dimGrid, dimBlock>>>(
        X->values_d,
        linear_weights->values_d,
        Y_d,
        num_samples,
        in_features, out_features
    );

    // Update tensor.
    X->dim[1] = out_features;
    hipFree(X->values_d);
    X->values_d = Y_d;
}


/**
 * Perform softmax function on a 2D tensor across column.
 * TODO: enable performing the softmax on n-dimensional tensor given the input axis.
 * 
 */
void run_log_softmax_forward(Tensor *tensor) {
    if (tensor->num_dim != 2) {
        printf("The input tensor must have 2 dimensions to perform softmax function.\n");
        free_tensor(tensor);
        tensor = NULL;
        return;
    }

    uint32_t num_samples  = tensor->dim[0];
    uint32_t num_features = tensor->dim[1];
    uint32_t out_size     = num_samples * num_features;

    float *X_output_d, *X_exp_sum_d;
    hipMalloc((void**)&X_output_d, out_size * sizeof(float));
    hipMalloc((void**)&X_exp_sum_d, num_samples * sizeof(float));
    hipMemset(X_exp_sum_d, 0, num_samples * sizeof(float));

    // TODO: consider the cases where the total size is significantly lower
    // than TILE_WIDTH * TILE_WIDTH;
    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);
    dim3 dimGrid(ceil(num_features * 1.0 / TILE_WIDTH), ceil(num_samples * 1.0 / TILE_WIDTH));
    CalcExpAndSumByRowKernel<<<dimGrid, dimBlock>>>(
        tensor->values_d, X_output_d, X_exp_sum_d, num_samples, num_features
    );

    LogNormalizeForwardKernel<<<dimGrid, dimBlock>>>(X_output_d, X_exp_sum_d, num_samples, num_features);

    hipFree(X_exp_sum_d);
    
    // Update tensor.
    hipFree(tensor->values_d);
    tensor->values_d = X_output_d;
}


Tensor *compute_negative_log_likelihood_lost(Tensor *tensor, uint8_t *y_d) {
    uint32_t num_samples = tensor->dim[0];

    float *out;
    hipMalloc((void**)&out, sizeof(float));
    hipMemset(out, 0, sizeof(float));

    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);
    dim3 dimGrid(ceil(LABEL_SIZE * 1.0 / (TILE_WIDTH * THREAD_COARSENING_FACTOR)), ceil(num_samples * 1.0 / TILE_WIDTH));
    NegativeLogLikelihoodKernel<<<dimGrid, dimBlock>>>(tensor->values_d, y_d, out, num_samples);

    Tensor *output = (Tensor *)malloc(sizeof(Tensor));
    output->num_dim = 1;
    uint32_t *dim = (uint32_t *)malloc(sizeof(uint32_t));
    dim[0] = 1;
    output->dim = dim;
    output->values_d = out;
    
    return output;
}
