#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdint.h>

#include "kernel_functions.cuh"


/**
 * (Not optimized) Conv2 kernel implementation, following the method in chapter 16.3 (Fig. 16.13,14).
 */
__global__ void Conv2ForwardKernel(
    float *X, float *Y,
    float *filters,
    uint32_t kernel_length,
    uint32_t in_channels,
    uint32_t grid_height, uint32_t grid_width,
    uint32_t in_height, uint32_t in_width,
    uint32_t out_height, uint32_t out_width
) {
    uint32_t out_channel_idx = blockIdx.x;
    uint32_t out_height_idx  = (blockIdx.y / grid_width)*TILE_WIDTH + threadIdx.y;
    uint32_t out_width_idx   = (blockIdx.y % grid_width)*TILE_WIDTH + threadIdx.x;
    uint32_t sample_idx      = blockIdx.z;
    uint32_t out_channels = gridDim.x;

    if (out_height_idx >= out_height || out_width_idx >= out_width)
        return;
    
    float value = 0.0f;
    for (uint32_t in_channel_idx = 0; in_channel_idx < in_channels; ++in_channel_idx)
        for (uint32_t k_row = 0; k_row < kernel_length; ++k_row)
            for (uint32_t k_col = 0; k_col < kernel_length; ++k_col) {
                uint32_t in_row = out_height_idx + k_row;
                uint32_t in_col = out_width_idx + k_col;
                
                uint32_t X_idx = (sample_idx * in_channels * in_height * in_width) + 
                    (in_channel_idx * in_height * in_width) + 
                    (in_row * in_width) + 
                    in_col;
                uint32_t weight_idx = (out_channel_idx * in_channels * kernel_length * kernel_length) +
                    (in_channel_idx * kernel_length * kernel_length) +
                    (k_row * kernel_length) +
                    k_col;
                value += X[X_idx] * filters[weight_idx];
            }
    
    uint32_t Y_idx = (sample_idx * out_channels * out_height * out_width) + 
            (out_channel_idx * out_height * out_width) +
            (out_height_idx * out_width) +
            out_width_idx;
    Y[Y_idx] = value;
}


__global__ void SigmoidForwardKernel(
    float *X, float *Y,
    uint32_t grid_height, uint32_t grid_width,
    uint32_t out_height, uint32_t out_width
) {
    uint32_t out_channel_idx = blockIdx.x;
    uint32_t out_height_idx  = (blockIdx.y / grid_width)*TILE_WIDTH + threadIdx.y;
    uint32_t out_width_idx   = (blockIdx.y % grid_width)*TILE_WIDTH + threadIdx.x;
    uint32_t sample_idx      = blockIdx.z;
    uint32_t num_channels = gridDim.x;

    if (out_height_idx >= out_height || out_width_idx >= out_width)
        return;

    for (uint32_t row = 0; row < out_height; ++row)
        for (uint32_t col = 0; col < out_width; ++col) {
            uint32_t index = (sample_idx * num_channels * out_height * out_width) +
                (out_channel_idx * out_height * out_width) +
                (row * out_width) +
                col;
            Y[index] = 1.0 / (1 + expf(-1 * X[index]));
        }
}


/**
 * Perform either max or mean pooling forward layer.
 * 
 * For now, assume that the stride is always kernel_length and the input width & height
 * are always divisible by kernel_length.
 */
__global__ void PoolForwardKernel(
    float *X, float *Y,
    pooling_type pool_type,
    uint32_t kernel_length,
    uint32_t grid_height, uint32_t grid_width,
    uint32_t in_height, uint32_t in_width,
    uint32_t out_height, uint32_t out_width
) {
    uint32_t num_channel_idx = blockIdx.x;
    uint32_t out_height_idx  = (blockIdx.y / grid_width)*TILE_WIDTH + threadIdx.y;
    uint32_t out_width_idx   = (blockIdx.y % grid_width)*TILE_WIDTH + threadIdx.x;
    uint32_t sample_idx      = blockIdx.z;
    uint32_t num_channels = gridDim.x;

    if (out_height_idx >= out_height || out_width_idx >= out_width)
        return;
    
    float value = 0.0f;
    for (uint32_t k_row = 0; k_row < kernel_length; ++k_row)
        for (uint32_t k_col = 0; k_col < kernel_length; ++k_col) {
            uint32_t in_row = kernel_length * out_height_idx + k_row;
            uint32_t in_col = kernel_length * out_width_idx + k_col;
            
            uint32_t X_idx = (sample_idx * num_channels * in_height * in_width) + 
                (num_channel_idx * in_height * in_width) + 
                (in_row * in_width) + 
                in_col;
            
            if (pool_type == MAX) {
                value = max(value, X[X_idx]);
            } else {
                value += (X[X_idx] / (kernel_length * kernel_length));
            }
        }

    uint32_t Y_idx = (sample_idx * num_channels * out_height * out_width) + 
            (num_channel_idx * out_height * out_width) +
            (out_height_idx * out_width) +
            out_width_idx;
    Y[Y_idx] = value;
}
