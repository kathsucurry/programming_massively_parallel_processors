#include "hip/hip_runtime.h"
/**
 * TODO:
 * - Load the entire MNIST data at once instead of per sample
 * - When changing the size of the image, avoid changing the image raw data
 * - Ensure all malloc and cudamalloc are successful (i.e., check for any errors)
 */

#include <stdio.h>
#include <stdint.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

#include "src/data_loader.cuh"
#include "src/preprocessing.cuh"
#include "src/common.h"
#include "src/cnn_layers.cuh"


// Assume fixed LR for now.
#define LEARNING_RATE 0.005


void eval_model() {

}


/**
 * For simplicity, the model architecture is fixed for now.
 * 
 * The model architecture is as follows.
 * - Convolution layer: 16 filters, each 5 x 5 --> output dim = 5 x 28 x 28
 * - Sigmoid layer
 * - Pooling layer: kernel size 2 x 2 --> output dim = 14 x 14
 * - Flatten layer --> output dim = 3136
 * - Linear layer (3136 x 10)
 * - Softmax layer
 * 
 */
NetworkOutputs *forward_pass(
    float *X_d, uint8_t *y_d,
    NetworkWeights *network_weights_d,
    uint32_t image_height,
    uint32_t image_width,
    uint32_t num_samples
) {
    uint8_t num_layers_with_grads = 5;
    LayerGradients *gradients = (LayerGradients *)malloc(num_layers_with_grads * sizeof(LayerGradients));
    
    uint32_t *image_dim = (uint32_t *)malloc(3 * sizeof(uint32_t));
    Tensor *output = initialize_tensor(X_d, 3, image_dim);
    
    // Layer 0: 2D convolution layer.
    run_conv2d_forward(output, network_weights_d->conv2d_weight, num_samples, image_height, image_width, &gradients[0]);
    
    // Layer 1: Sigmoid activation.
    run_sigmoid_forward(output, &gradients[1]);

    // Layer 2: Max pooling layer.
    uint32_t pool_kernel_length = 2;
    pooling_type pool_type = MAX;
    run_pooling_forward(output, pool_kernel_length, pool_type, &gradients[2]);
    
    // Layer 3: Convert into 1D vector; no grads.
    run_flatten_layer(output);

    // Layer 4: Linear layer.
    run_linear_forward(output, network_weights_d->linear_weight, &gradients[3]);

    // Layer 5: Softmax layer.
    run_softmax_forward(output, y_d, &gradients[4]);

    NetworkOutputs *network_outputs = (NetworkOutputs *)malloc(sizeof(NetworkOutputs));
    network_outputs->gradients = gradients;
    network_outputs->output = output;

    return network_outputs;
}


void backward_pass(NetworkOutputs *network_outputs, NetworkWeights *network_weights) {
    // Update linear weights.
    // Update conv2d weights.
}

NetworkWeights *train_model(ImageDataset *dataset, uint32_t batch_size) {
    // uint32_t num_samples = dataset->num_samples;
    uint32_t num_samples = dataset->num_samples;

    // Perform simple dataset split into training and validation.
    // Another approach is to assess the label distribution and split based on that
    // assuming the test data distribution follows the training data distribution.
    uint32_t num_train_samples = DATASET_SPLIT_TRAIN_PROPORTION * num_samples;
    uint32_t num_valid_samples = (1 - DATASET_SPLIT_TRAIN_PROPORTION) * num_samples;

    ImageDataset *train = split_dataset(dataset, 0, num_train_samples);
    ImageDataset *valid = split_dataset(dataset, num_train_samples, num_train_samples + num_valid_samples);
    if (train == NULL || valid == NULL) {
        printf("Error in dataset split.");
        return NULL;
    }

    // Prepare the model architecture: conv -> sigmoid -> pooling -> flatten -> linear -> softmax.
    // Initialize conv and linear layer weights using device memory.
    NetworkWeights *network_weights = (NetworkWeights *)malloc(sizeof(NetworkWeights));
    network_weights->conv2d_weight = initialize_conv_layer_weights(1, 16, 5, 0);
    network_weights->linear_weight = initialize_linear_layer_weights(3136, 10, 1);

    uint32_t num_epochs = 5;
    uint32_t num_epochs_valid_iter = 2;
    uint32_t num_batches = ceil(num_train_samples * 1.0 / BATCH_SIZE);
    uint32_t image_height = train->images[0].height;
    uint32_t image_width = train->images[0].width;
    uint32_t image_size = image_height * image_width;
    
    float train_X[BATCH_SIZE * image_size];
    uint8_t train_y[BATCH_SIZE * LABEL_SIZE];

    float *train_X_d;
    uint8_t *train_y_d;

    hipMalloc((void**)&train_X_d, BATCH_SIZE * image_size * sizeof(float));
    hipMalloc((void**)&train_y_d, BATCH_SIZE * LABEL_SIZE * sizeof(uint8_t));

    // For each epoch, run forward pass, evaluate on validation (i.e., forward pass + assess), backward pass.
    for (uint32_t epoch_index = 0; epoch_index < num_epochs; ++epoch_index) {
        // Shuffle the training indices.
        shuffle_indices(train, epoch_index);
        for (uint32_t batch_index = 0; batch_index < num_batches; ++batch_index) {
            uint32_t num_samples_in_batch = min(num_train_samples - batch_index * BATCH_SIZE, BATCH_SIZE);
            
            // Fill the batch.
            prepare_batch(train_X, train_y, train, num_samples_in_batch);

            // Copy host variables to device memory.
            hipMemcpy(train_X_d, train_X, num_samples_in_batch * image_size * sizeof(float), hipMemcpyHostToDevice);
            hipMemcpy(train_y_d, train_y, num_samples_in_batch * sizeof(uint8_t), hipMemcpyHostToDevice);

            NetworkOutputs *network_outputs = forward_pass(train_X_d, train_y_d, network_weights, image_height, image_width, num_samples_in_batch);
            Tensor *loss = compute_negative_log_likelihood_log_lost(network_outputs->output, train_y_d);
            backward_pass(network_outputs, network_weights);
            
        }
        break;
        if (epoch_index > 0 && epoch_index % num_epochs_valid_iter == 0) {
            // float *valid_X[BATCH_SIZE * image_size];
            // uint8_t valid_y[BATCH_SIZE];
            // float *valid_logits = forward_pass(valid_X, valid_y, network_weights, image_size, num_valid_samples);
            // Calculate loss.
            // Evaluate model.
        }
    }

    hipFree(train_X_d);
    hipFree(train_y_d);

    free_image_dataset(train);
    free_image_dataset(valid);

    return network_weights;
}


int main() {
    MNISTDataset *train_dataset = load_mnist_dataset(
        "../../Dataset/mnist/train-images-idx3-ubyte",
        "../../Dataset/mnist/train-labels-idx1-ubyte"
    );
    MNISTDataset *test_dataset = load_mnist_dataset(
        "../../Dataset/mnist/t10k-images-idx3-ubyte",
        "../../Dataset/mnist/t10k-labels-idx1-ubyte"
    );

    printf("# Samples in training set: %d\n", train_dataset->num_samples);
    printf("# Samples in test set: %d\n", test_dataset->num_samples);

    // Normalize the pixel values to [0..1].
    ImageDataset *transformed_train_dataset = add_padding(
        normalize_pixels(
            prepare_dataset(train_dataset)
        ),
        2
    );

    NetworkWeights *model_weights = train_model(transformed_train_dataset, BATCH_SIZE);

    // Run evaluation on the test set.

    free(model_weights);
}