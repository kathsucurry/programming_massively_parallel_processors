/**
 * Performs small tests with the following steps:
 * 1) Load and store only the first five images.
 * 2) Run 1 batch (size = 3) iteration of forward and backward pass.
 * 3) Check results along the way.
 * 
 * The model architecture is identical to the one used in train_mnist.cu:
 * - Convolution layer: 4 filters, each 5 x 5 --> output dim = 4 x 28 x 28
 * - Sigmoid layer
 * - Pooling layer: kernel size 2 x 2 --> output dim = 14 x 14
 * - Flatten layer --> output dim = 784 (4 filters x 14 x 14)
 * - Linear layer (784 x 10)
 * - Softmax layer
 * 
 */

#include <stdio.h>
#include <stdint.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

#include "../src/data_loader.cuh"
#include "../src/preprocessing.cuh"
#include "../src/common.cuh"
#include "../src/cnn_layers.cuh"
#include "test_utils.cuh"
#include "test_data_prep.cuh"


#define LEARNING_RATE 0.005
#define POOL_KERNEL_LENGTH 2
#define NUM_SAMPLES 5


NetworkOutputs *forward_pass(
    float *X_d, uint8_t *y_d,
    NetworkWeights *network_weights_d,
    uint32_t image_height,
    uint32_t image_width,
    uint32_t num_samples,
    bool compute_grad
) {
    uint8_t num_layers_with_grads = 6;
    LayerGradients *gradients = (LayerGradients *)mallocCheck(num_layers_with_grads * sizeof(LayerGradients));
    
    uint32_t *image_dim = (uint32_t *)mallocCheck(4 * sizeof(uint32_t));
    image_dim[0] = num_samples;
    image_dim[1] = 1; // Number of channels.
    image_dim[2] = image_height;
    image_dim[3] = image_width;
    Tensor *output = initialize_tensor(X_d, 4, image_dim);
    
    // Layer 0: 2D convolution layer.
    run_conv2d_forward(output, network_weights_d->conv2d_weight, &gradients[0], compute_grad);
    compare_tensor("layer 0 conv2d", "tests/outputs/output_layer0_conv2d.txt", output);

    // Layer 1: Sigmoid activation.
    run_sigmoid_forward(output, &gradients[1], compute_grad);
    compare_tensor("layer 1 sigmoid", "tests/outputs/output_layer1_sigmoid.txt", output);

    // Layer 2: Max pooling layer.
    uint32_t pool_kernel_length = POOL_KERNEL_LENGTH;
    pooling_type pool_type = MAX;
    run_pooling_forward(output, pool_kernel_length, pool_type, &gradients[2], compute_grad);
    compare_tensor("layer 2 maxpool", "tests/outputs/output_layer2_maxpool.txt", output);
    
    // Layer 3: Convert into 1D vector; no grads created.
    run_flatten_forward(output);
    compare_tensor("layer 3 flatten", "tests/outputs/output_layer3_flatten.txt", output);

    // Layer 4: Linear layer.
    run_linear_forward(output, network_weights_d->linear_weight, &gradients[4], compute_grad);
    compare_tensor("layer 4 linear", "tests/outputs/output_layer4_linear.txt", output);

    // Layer 5: Softmax layer.
    run_softmax_forward(output, y_d, &gradients[5], compute_grad);
    compare_tensor("layer 5 softmax", "tests/outputs/output_layer5_softmax.txt", output);

    NetworkOutputs *network_outputs = (NetworkOutputs *)mallocCheck(sizeof(NetworkOutputs));
    network_outputs->gradients = gradients;
    network_outputs->output = output;
    network_outputs->num_layers = 6;

    return network_outputs;
}


void backward_pass(LayerGradients *gradients, NetworkWeights *network_weights, uint32_t num_samples, float learning_rate) {
    // Go through layers from the second last to the first to update gradients + weights.
    compare_tensor("layer 4 linear dY", "tests/outputs/dy_layer4_linear.txt", gradients[5].dX_or_X);
    
    // Layer 4: linear layer - update both gradients and weights.
    run_linear_backward(network_weights->linear_weight, &gradients[4], &gradients[5], learning_rate);
    compare_tensor("layer 4 linear dX", "tests/outputs/dy_layer3_flatten.txt", gradients[4].dX_or_X);
    compare_tensor("layer 4 linear dW", "tests/outputs/weight_grad_layer4_linear.txt", gradients[4].dW_or_W);
    compare_tensor("layer 4 linear updated W", "tests/outputs/updated_weight_layer4_linear.txt", network_weights->linear_weight);

    // Layer 3: flatten layer (i.e., change the dimension of the next layer's gradients).
    run_flatten_backward(num_samples, POOL_KERNEL_LENGTH, &gradients[3], &gradients[4]);
    compare_tensor("layer 3 flatten dX", "tests/outputs/dy_layer2_maxpool.txt", gradients[3].dX_or_X);

    // Layer 2: pooling layer.
    run_pooling_backward(POOL_KERNEL_LENGTH, &gradients[2], &gradients[3]);
    compare_tensor("layer 2 maxpool dX", "tests/outputs/dy_layer1_sigmoid.txt", gradients[2].dX_or_X);

    // Layer 1: sigmoid layer.
    run_sigmoid_backward(&gradients[1], &gradients[2]);
    compare_tensor("layer 1 sigmoid dX", "tests/outputs/dy_layer0_conv2d.txt", gradients[1].dX_or_X);
    
    // Layer 0: conv2d layer - update both gradients and weights.
    run_conv2d_backward(network_weights->conv2d_weight, &gradients[0], &gradients[1], learning_rate);
    compare_tensor("layer 0 conv2d dW", "tests/outputs/weight_grad_layer0_conv2d.txt", gradients[0].dW_or_W);
    compare_tensor("layer 0 conv2d updated W", "tests/outputs/updated_weight_layer0_conv2d.txt", network_weights->conv2d_weight);
}

NetworkWeights *train_model(ImageDataset *dataset, uint32_t batch_size) {
    // Prepare the model architecture: conv -> sigmoid -> pooling -> flatten -> linear -> softmax.
    // Initialize conv and linear layer weights using device memory.
    NetworkWeights *network_weights = (NetworkWeights *)mallocCheck(sizeof(NetworkWeights));
    network_weights->conv2d_weight = initialize_conv_layer_weights(1, 4, 5, 0);
    test_initialize_conv_layer_weights(network_weights->conv2d_weight);
    
    network_weights->linear_weight = initialize_linear_layer_weights(784, 10, 1);
    test_initialize_linear_layer_weights(network_weights->linear_weight);

    uint32_t image_height = dataset->images[0].height;
    uint32_t image_width = dataset->images[0].width;
    uint32_t image_size = image_height * image_width;
    
    float train_X[batch_size * image_size];
    uint8_t train_y[batch_size * LABEL_SIZE];

    float *train_X_d;
    uint8_t *train_y_d;

    cudaMallocCheck((void**)&train_X_d, batch_size * image_size * sizeof(float));
    cudaMallocCheck((void**)&train_y_d, batch_size * LABEL_SIZE * sizeof(uint8_t));

    shuffle_indices(dataset, 0);
    test_shuffle_indices(dataset);

    // Fill the batch.
    prepare_batch(train_X, train_y, dataset, 0, batch_size);
    test_prepare_batch(train_X, train_y, dataset, false);

    // Copy host variables to device memory.
    hipMemcpy(train_X_d, train_X, batch_size * image_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(train_y_d, train_y, batch_size * LABEL_SIZE * sizeof(uint8_t), hipMemcpyHostToDevice);

    NetworkOutputs *network_outputs = forward_pass(train_X_d, train_y_d, network_weights, image_height, image_width, batch_size, true);
    
    float *loss = compute_negative_log_likelihood_log_lost(network_outputs->output, train_y_d);
    compare_float("cross entropy loss", "tests/outputs/output_loss.txt", loss);

    backward_pass(network_outputs->gradients, network_weights, batch_size, LEARNING_RATE);

    free(loss);
    free_network_outputs(network_outputs, true);
    hipFree(train_X_d);
    hipFree(train_y_d);

    return network_weights;
}


int main() {
    MNISTDataset *dataset = load_mnist_dataset(
        "../../Dataset/mnist/train-images-idx3-ubyte",
        "../../Dataset/mnist/train-labels-idx1-ubyte"
    );

    // Keep only the first three samples.
    keep_first_n_samples(dataset, NUM_SAMPLES);

    printf("# Samples in training set: %d\n", dataset->num_samples);

    // Normalize the pixel values to [0..1].
    ImageDataset *transformed_train_dataset = add_padding(
        normalize_pixels(
            prepare_dataset(dataset)
        ),
        2
    );

    free_MNIST_dataset(dataset);

    // print_sample(transformed_train_dataset, 2);

    NetworkWeights *model_weights = train_model(transformed_train_dataset, 3);

    // Run evaluation on the test set.

    free_dataset(transformed_train_dataset);
    free_network_weights(model_weights);
}