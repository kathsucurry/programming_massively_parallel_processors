#include "hip/hip_runtime.h"
/**
 * Perform mini test for the training. TODO: write unit tests.
 */

#include <stdio.h>
#include <stdint.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

#include "src/data_loader.cuh"
#include "src/preprocessing.cuh"
#include "src/common.h"
#include "src/cnn_layers.cuh"


void print_tensor_2d(Tensor *tensor) {
    uint32_t out_size = get_tensor_values_size(tensor->num_dim, tensor->dim);
    float *values = (float *)malloc(out_size * sizeof(float));
    hipMemcpy(values, tensor->values_d, out_size * sizeof(float), hipMemcpyDeviceToHost);
    uint32_t *dim = tensor->dim;

    for (uint32_t row = 0; row < dim[0]; ++row) {
        for (uint32_t col = 0; col < dim[1]; ++col)
            printf("%10.2f", values[row * dim[1] + col]);
        printf("\n");
    }

    free(values);
}


void print_tensor_4d(Tensor *tensor) {
    uint32_t out_size = get_tensor_values_size(tensor->num_dim, tensor->dim);
    float *values = (float *)malloc(out_size * sizeof(float));
    hipMemcpy(values, tensor->values_d, out_size * sizeof(float), hipMemcpyDeviceToHost);
    uint32_t *dim = tensor->dim;

    for (uint32_t sample_index = 0; sample_index < dim[0]; ++sample_index) {
        printf("Sample index %u\n", sample_index);
        for (uint32_t channel_index = 0; channel_index < dim[1]; ++channel_index) {
            printf("Channel index %u\n", channel_index);
            for (uint32_t row = 0; row < dim[2]; ++row) {
                for (uint32_t col = 0; col < dim[3]; ++col)
                    printf("%6.2f", values[sample_index * out_size / dim[0] + channel_index * dim[2] * dim[3] + row * dim[3] + col]);
                printf("\n");
            }
            printf("\n");
        }
        printf("\n");
    }

    free(values);
}


Tensor *generate_test_ordered_tensor_2d(uint32_t height, uint32_t width, float multiplier) {
    Tensor *tensor = (Tensor *)malloc(sizeof(Tensor));
    tensor->num_dim = 2;
    uint32_t *dim = (uint32_t *)malloc(2 * sizeof(uint32_t));
    dim[0] = height;
    dim[1] = width;
    tensor->dim = dim;
    
    float counter = 0.0f;
    float *values = (float *)malloc(height * width * sizeof(float));
    for (uint32_t row = 0; row < height; ++row) {
        for (uint32_t col = 0; col < width; ++col) {
            uint32_t index = row * width + col;
            values[index] = multiplier * counter++;
            printf("%8.3f", values[index]);
        }
        printf("\n");
    }
    printf("\n");
    
    float *values_d;
    hipMalloc((void**)&values_d, height * width * sizeof(float));
    hipMemcpy(values_d, values, height * width * sizeof(float), hipMemcpyHostToDevice);
    tensor->values_d = values_d;

    free(values);
    return tensor;
}


Tensor *generate_test_random_4d_tensor(uint32_t *dim, float multiplier, int seed) {
    srand(seed);

    Tensor *tensor = (Tensor *)malloc(sizeof(Tensor));
    tensor->num_dim = 4;
    
    uint32_t *tensor_dim = (uint32_t *)malloc(4 * sizeof(uint32_t));
    memcpy(tensor_dim, dim, 4 * sizeof(uint32_t));
    tensor->dim = tensor_dim;
    
    uint32_t out_size = get_tensor_values_size(4, dim);
    float *values = (float *)malloc(out_size * sizeof(float));
    for (uint32_t i = 0; i < out_size; ++i)
        values[i] = multiplier * (rand() % 50);
    
    float *values_d;
    hipMalloc((void**)&values_d, out_size * sizeof(float));
    hipMemcpy(values_d, values, out_size * sizeof(float), hipMemcpyHostToDevice);
    tensor->values_d = values_d;
    free(values);
    return tensor;
}


void test_transpose_matrix() {
    printf("--> Test transpose matrix...\n");
    uint32_t input_height = 9;
    uint32_t input_width  = 7;
    printf("Input:\n");
    Tensor *input = generate_test_ordered_tensor_2d(input_height, input_width, 1);

    float *output_d;
    hipMalloc((void**)&output_d, input_height * input_width * sizeof(float));
    hipMemset(output_d, 0, input_height * input_width * sizeof(float));

    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);
    dim3 dimGrid(ceil(input_height * 1.0 / TILE_WIDTH), ceil(input_width * 1.0 / TILE_WIDTH));
    TransposeMatrixKernel<<<dimGrid, dimBlock>>>(input->values_d, output_d, input_width, input_height);

    Tensor *output = (Tensor *)malloc(sizeof(Tensor));
    output->num_dim = 2;
    output->dim = (uint32_t *)malloc(2 * sizeof(uint32_t));
    output->dim[0] = input_width;
    output->dim[1] = input_height;
    output->values_d = output_d;

    printf("Output:\n");
    print_tensor_2d(output);
    printf("\n");

    free_tensor(input);
    free_tensor(output);
}


void test_matmul() {
    printf("--> Test matrix multiplication...\n");
    uint32_t input1_height = 9;
    uint32_t input1_width  = 7;
    uint32_t input2_width = 5;
    printf("Input 1:\n");
    Tensor *input1 = generate_test_ordered_tensor_2d(input1_height, input1_width, 1);

    printf("Input 2:\n");
    Tensor *input2 = generate_test_ordered_tensor_2d(input1_width, input2_width, 1);

    float *output_d;
    hipMalloc((void**)&output_d, input1_height * input2_width * sizeof(float));
    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);
    dim3 dimGrid(ceil(input2_width * 1.0 / TILE_WIDTH / THREAD_COARSENING_FACTOR), ceil(input1_height * 1.0 / TILE_WIDTH));
    MatMulKernel<<<dimGrid, dimBlock>>>(input1->values_d, input2->values_d, output_d, input1_height, input1_width, input2_width);

    Tensor *output = (Tensor *)malloc(sizeof(Tensor));
    output->num_dim = 2;
    output->dim = (uint32_t *)malloc(2 * sizeof(uint32_t));
    output->dim[0] = input1_height;
    output->dim[1] = input2_width;
    output->values_d = output_d;

    printf("Output:\n");
    print_tensor_2d(output);
    printf("\n");

    free_tensor(input1);
    free_tensor(input2);
    free_tensor(output);
}


void print_conv2d_weight_init_example(uint32_t in_channels, uint32_t out_channels, uint8_t filter_size) {
    printf("--> Assess the conv layer weight initialization...");
    
    Tensor *conv2d_weight = initialize_conv_layer_weights(1, 3, 3, 3);
    printf("Conv2 weight has %u dimensions: ", conv2d_weight->num_dim);

    for (uint8_t i = 0; i < conv2d_weight->num_dim; ++i)
        printf("%u ", conv2d_weight->dim[i]);
    printf("\n");
    uint32_t weight_size = get_tensor_values_size(conv2d_weight->num_dim, conv2d_weight->dim);
    printf("Weight size: %u\n", weight_size);

    printf("Weight values: ");
    float *weights = (float *)malloc(weight_size * sizeof(float));
    hipMemcpy(weights, conv2d_weight->values_d, weight_size * sizeof(float), hipMemcpyDeviceToHost);

    for (uint32_t i = 0; i < weight_size; ++i)
        printf("%.2f ", weights[i]);
    printf("\n-------\n\n");

    free(weights);
    free_tensor(conv2d_weight);
}


/**
 * Generate a tiny dataset where all images are the same with float pixel values [0..image_height * image_width);
 */
ImageDataset *generate_tiny_dataset(uint32_t num_samples, uint32_t image_height, uint32_t image_width) {
    printf("--> Generate a tiny dataset with %u samples and images of [%u x %u]...\n", num_samples, image_height, image_width);

    ImageDataset *dataset = (ImageDataset *)malloc(sizeof(ImageDataset));
    Image *images = (Image *)malloc(num_samples * sizeof(Image));
    uint32_t *view_indices = (uint32_t *)malloc(num_samples * sizeof(uint32_t));
    for (uint8_t image_index = 0; image_index < num_samples; ++image_index) {
        view_indices[image_index] = image_index;
        float *pixels = (float *)malloc(image_height * image_width * sizeof(float));
        for (uint8_t i = 0; i < image_height * image_width; ++i)
            pixels[i] = i;
        images[image_index].height = image_height;
        images[image_index].width = image_width;
        images[image_index].pixels = pixels;
    }
    dataset->images = images;
    dataset->labels = NULL;
    dataset->num_samples = num_samples;
    dataset->view_indices = view_indices;

    printf("Image example:\n");
    Image image = dataset->images[0];
    for (uint32_t row = 0; row < image.width; ++row) {
        for (uint32_t col = 0; col < image.width; ++col) {
            uint32_t index = row * image.width + col;
            printf("%3.0f", image.pixels[index]);
        }
        printf("\n");
    }
    printf("\n");
    return dataset;
}


Tensor *generate_custom_weights(uint32_t in_channels, uint32_t out_channels, uint8_t kernel_length) {
    printf("--> Generate a custom weights...\n");
    
    Tensor *weights = (Tensor *)malloc(sizeof(Tensor));
    
    weights->num_dim = 4;
    uint32_t *dim = (uint32_t *)malloc(4 * sizeof(uint32_t));
    dim[0] = out_channels;
    dim[1] = in_channels;
    dim[2] = kernel_length;
    dim[3] = kernel_length;
    weights->dim = dim;

    printf("Conv2 weight to be used for further testing has %u dimensions: ", weights->num_dim);
    for (uint8_t i = 0; i < weights->num_dim; ++i)
        printf("%u ", dim[i]);
    printf("\n");

    uint32_t weight_size = get_tensor_values_size(weights->num_dim, dim);

    float *values = (float *)malloc(weight_size * sizeof(float));
    for (uint32_t filter_index = 0; filter_index < out_channels; ++filter_index) {
        float counter = 0.0f;
        for (uint32_t row = 0; row < kernel_length; ++row)
            for (uint32_t col = 0; col < kernel_length; ++col) {
                uint32_t index = filter_index * kernel_length * kernel_length + row * kernel_length + col;
                values[index] = counter++;
            }
    }
    printf("Weight values: ");
    for (uint32_t i = 0; i < weight_size; ++i)
        printf("%.0f ", values[i]);
    printf("\n\n");

    float *values_d;
    hipMalloc((void**)&values_d, weight_size * sizeof(float));
    hipMemcpy(values_d, values, weight_size * sizeof(float), hipMemcpyHostToDevice);
    weights->values_d = values_d;

    return weights;
}


float *generate_X_device(ImageDataset *dataset) {
    uint32_t image_height = dataset->images[0].height;
    uint32_t image_width = dataset->images[0].width;
    float *X = (float *)malloc(dataset->num_samples * image_height * image_width * sizeof(float));
    for (uint32_t sample_index = 0; sample_index < dataset->num_samples; ++sample_index) {
        uint32_t sample_offset = sample_index * image_height * image_width;
        Image image = dataset->images[sample_index];
        for (uint32_t row = 0; row < image_height; ++row)
            for (uint32_t col = 0; col < image_width; ++col) {
                X[sample_offset + row * image_width + col] = image.pixels[row * image_width + col];
            }
    }
    
    float *X_d;
    hipMalloc((void**)&X_d, dataset->num_samples * image_height * image_width * sizeof(float));
    hipMemcpy(X_d, X, dataset->num_samples * image_height * image_width * sizeof(float), hipMemcpyHostToDevice);

    free(X);
    return X_d;
}


void run_conv2d_forward_test() {
    printf("--> Perform conv2d test...\n");

    // Create a small ImageDataset; each image is only 5 x 5.
    uint32_t num_samples = 3;
    uint32_t image_height = 5;
    uint32_t image_width = 5;
    ImageDataset *dataset = generate_tiny_dataset(num_samples, image_height, image_width);
    float *X_d = generate_X_device(dataset);

    // Generate custom weights.
    uint32_t kernel_length = 3;
    uint32_t num_kernels = 3;
    uint32_t in_channels = 1;
    uint32_t show_sample_index = 0;
    Tensor *conv2d_weight = generate_custom_weights(in_channels, num_kernels, kernel_length);

    uint32_t *image_dim = (uint32_t *)malloc(3 * sizeof(uint32_t));
    image_dim[0] = num_samples;
    image_dim[1] = image_height;
    image_dim[2] = image_width;
    Tensor *output = initialize_tensor(X_d, 3, image_dim);
    LayerGradients *grad = (LayerGradients *)malloc(sizeof(LayerGradients));
    run_conv2d_forward(output, conv2d_weight, num_samples, image_height, image_width, grad);
    
    printf("Output description:\n");
    printf("# Dim: %u [", output->num_dim);
    for (uint8_t i = 0; i < output->num_dim; ++i)
        printf("%2u", output->dim[i]);
    printf("]\n\n");

    printf("Printing sample index %u:\n\n", show_sample_index);
    uint32_t sample_size = get_tensor_values_size(output->num_dim, output->dim) / output->dim[0];

    float *values = (float *)malloc(sample_size * sizeof(float));
    hipMemcpy(values, &output->values_d[show_sample_index*sample_size], sample_size * sizeof(float), hipMemcpyDeviceToHost);

    uint32_t feature_map_size = output->dim[2] * output->dim[3];

    for (uint32_t feature_index = 0; feature_index < output->dim[1]; ++feature_index) {
        printf("Feature map %u:\n", feature_index);
        for (uint32_t row = 0; row < output->dim[2]; ++row) {
            for (uint32_t col = 0; col < output->dim[3]; ++col) {
                printf("%4.0f", values[feature_index * feature_map_size + row * output->dim[3] + col]);
            }
            printf("\n");
        }
    }
    printf("\n");

    free_tensor(output);
    free_tensor(conv2d_weight);
    hipFree(X_d);
    free_dataset(dataset);
}


void run_sigmoid_forward_test() {
    printf("--> Perform sigmoid layer test...\n");
    printf("Input:\n\n");

    uint8_t num_dim = 4;
    uint32_t *dim = (uint32_t *)malloc(num_dim * sizeof(uint32_t));
    dim[0] = 2; // # samples.
    dim[1] = 1; // # channels.
    dim[2] = 8; // Height.
    dim[3] = 6; // Width.
    Tensor *tensor = generate_test_random_4d_tensor(dim, 0.08, 0);
    print_tensor_4d(tensor);

    LayerGradients *grad = (LayerGradients *)malloc(sizeof(LayerGradients));

    run_sigmoid_forward(tensor, grad);

    printf("Sigmoid output:\n");
    print_tensor_4d(tensor);

    printf("Gradient:\n");
    print_tensor_4d(grad->dX_or_X);

    free_tensor(tensor);
    free_layer_gradients(grad);
}


void run_pool_forward_test() {
    printf("--> Perform pooling layer test with kernel size [2, 2]...\n");
    printf("Input:\n\n");

    uint8_t num_dim = 4;
    uint32_t *dim = (uint32_t *)malloc(num_dim * sizeof(uint32_t));
    dim[0] = 2; // # samples.
    dim[1] = 1; // # channels.
    dim[2] = 8; // Height.
    dim[3] = 6; // Width.
    Tensor *tensor = generate_test_random_4d_tensor(dim, 1, 0);
    print_tensor_4d(tensor);

    LayerGradients *grad = (LayerGradients *)malloc(sizeof(LayerGradients));

    printf("Max pool output:\n\n");
    run_pooling_forward(tensor, 2, MAX, grad);
    
    printf("Layer output:\n");
    print_tensor_4d(tensor);
    
    printf("Gradient:\n");
    print_tensor_4d(grad->dX_or_X);
    free_tensor(tensor);
    
    tensor = generate_test_random_4d_tensor(dim, 1, 0);
    printf("Mean pool output:\n\n");
    run_pooling_forward(tensor, 2, MEAN, grad);
    
    printf("Layer output:\n");
    print_tensor_4d(tensor);
    
    printf("Gradient:\n");
    print_tensor_4d(grad->dX_or_X);
    
    free_tensor(tensor);
    free_layer_gradients(grad);
}


void run_linear_layer_test() {
    printf("--> Perform linear layer test...\n");

    uint32_t feature1_height = 4;
    uint32_t feature1_width  = 5;
    uint32_t feature2_width  = 7;

    // Prepare feature 1.
    printf("Generating X:\n");
    Tensor *X = generate_test_ordered_tensor_2d(feature1_height, feature1_width, 1);
    printf("Generating A:\n");
    Tensor *A = generate_test_ordered_tensor_2d(feature2_width, feature1_width, 1);

    // Recall that the output will be stored in feature1.
    LayerGradients *gradients = (LayerGradients *)malloc(sizeof(LayerGradients));
    run_linear_forward(X, A, gradients);
    free_tensor(A);

    uint32_t out_height = feature1_height;
    uint32_t out_width  = feature2_width;
    float *values = (float *)malloc(out_height * out_width * sizeof(float));
    hipMemcpy(values, X->values_d, out_height * out_width * sizeof(float), hipMemcpyDeviceToHost);
    free_tensor(X);

    printf("Output: \n");
    for (uint32_t row = 0; row < out_height; ++row) {
        for (uint32_t col = 0; col < out_width; ++col) {
            printf("%8.0f", values[row * out_width + col]);
        }
        printf("\n");
    }
    printf("\n");
    free(values);

    printf("Gradients:\n");
    printf("dW\n");
    uint32_t dW_size = get_tensor_values_size(gradients->dW_or_W->num_dim, gradients->dW_or_W->dim);
    float *dW_values = (float *)malloc(dW_size * sizeof(float));
    hipMemcpy(dW_values, gradients->dW_or_W->values_d, dW_size * sizeof(float), hipMemcpyDeviceToHost);
    for (uint32_t row = 0; row < gradients->dW_or_W->dim[0]; ++row) {
        for (uint32_t col = 0; col < gradients->dW_or_W->dim[1]; ++col) {
            printf("%10.2f", dW_values[row * gradients->dW_or_W->dim[1] + col]);
        }
        printf("\n");
    }

    printf("\ndX\n");
    uint32_t dX_size = get_tensor_values_size(gradients->dX_or_X->num_dim, gradients->dX_or_X->dim);
    float *dX_values = (float *)malloc(dX_size * sizeof(float));
    hipMemcpy(dX_values, gradients->dX_or_X->values_d, dX_size * sizeof(float), hipMemcpyDeviceToHost);
    for (uint32_t row = 0; row < gradients->dX_or_X->dim[0]; ++row) {
        for (uint32_t col = 0; col < gradients->dX_or_X->dim[1]; ++col) {
            printf("%8.3f", dX_values[row * gradients->dX_or_X->dim[1] + col]);
        }
        printf("\n");
    }
    printf("\n");
}


void run_softmax_and_negative_log_likelihood_loss_test() {
    printf("--> Perform softmax and nll loss test...\n");
    printf("Input (X):\n");

    // Build X.
    uint32_t num_samples  = 5;
    uint32_t num_labels = 10;
    Tensor *X = generate_test_ordered_tensor_2d(num_samples, num_labels, 0.01);

    // Build y.
    uint8_t *y = (uint8_t *)calloc(num_samples * num_labels, sizeof(uint8_t));
    for (uint32_t i = 0; i < num_samples; ++i)
        y[i * num_labels + i] = 1;
    printf("Input (labels):\n");
    for (uint32_t i = 0; i < num_samples; ++i) {
        for (uint8_t label = 0; label < num_labels; ++label)
            printf("%2u", y[i * num_labels + label]);
        printf("\n");
    }
    printf("\n");

    uint8_t *y_d;
    hipMalloc((void**)&y_d, num_samples * num_labels * sizeof(uint8_t));
    hipMemcpy(y_d, y, num_samples * num_labels * sizeof(uint8_t), hipMemcpyHostToDevice);
    free(y);

    // Run softmax.
    LayerGradients *gradients = (LayerGradients *)malloc(sizeof(LayerGradients));
    run_softmax_forward(X, y_d, gradients);

    float *values = (float *)malloc(num_samples * num_labels * sizeof(float));
    hipMemcpy(values, X->values_d, num_samples * num_labels * sizeof(float), hipMemcpyDeviceToHost);

    printf("Softmax results: \n");
    for (uint32_t row = 0; row < num_samples; ++row) {
        for (uint32_t col = 0; col < num_labels; ++col) {
            printf("%9.4f", values[row * num_labels + col]);
        }
        printf("\n");
    }
    printf("\n");
    free(values);

    // Run NLL loss.
    Tensor *loss = compute_negative_log_likelihood_log_lost(X, y_d);
    free_tensor(X);
    hipFree(y_d);
    
    float *loss_h = (float *)malloc(sizeof(float));
    hipMemcpy(loss_h, loss->values_d, sizeof(float), hipMemcpyDeviceToHost);
    printf("Loss output: %.3f\n\n", *loss_h);
    free(loss_h);

    // Print softmax gradients.
    float *dX = (float *)malloc(num_samples * num_labels * sizeof(float));
    hipMemcpy(dX, gradients->dX_or_X->values_d, num_samples * num_labels * sizeof(float), hipMemcpyDeviceToHost);
    printf("Softmax gradients: \n");
    for (uint32_t row = 0; row < num_samples; ++row) {
        for (uint32_t col = 0; col < num_labels; ++col) {
            printf("%8.3f", dX[row * num_labels + col]);
        }
        printf("\n");
    }
    printf("\n\n");
    free(dX);

    free_layer_gradients(gradients);
}


int main() {
    test_transpose_matrix();

    test_matmul();

    print_conv2d_weight_init_example(1, 3, 3);
    
    run_conv2d_forward_test();

    run_sigmoid_forward_test();

    run_pool_forward_test();

    run_linear_layer_test();

    run_softmax_and_negative_log_likelihood_loss_test();
}