#include <stdio.h>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 16
#define TILE_WIDTH 2


/**
 *  Performs a simple matrix multiplication with shared memory, assuming all matrices are square with the same Width.
 * 
 *  One example code can be found here: https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#shared-memory.
 */
__global__
void MatrixMultiplicationKernel(
    int* matrix_M,
    int* matrix_N,
    int* matrix_Out,
    int Width
) {
    // Initialize the shared memories.
    __shared__  float shared_M[TILE_WIDTH][TILE_WIDTH];
    __shared__  float shared_N[TILE_WIDTH][TILE_WIDTH];

    int block_x = blockIdx.x;
    int block_y = blockIdx.y;
    int thread_x = threadIdx.x;
    int thread_y = threadIdx.y;

    // Identify the row and column of the matrix_Out matrix to work on.
    int Row = block_y * TILE_WIDTH + thread_y;
    int Col = block_x * TILE_WIDTH + thread_x;

    if ((Row < Width) && (Col < Width)) {
        // Loop over the tiles required to compute matrix_Out elements.
        int out_value = 0;
        for (int phase = 0; phase < Width/TILE_WIDTH; ++phase) {
            // Collaboratively load M and N tiles into shared memory.
            shared_M[thread_y][thread_x] = matrix_M[Row*Width + phase*TILE_WIDTH + thread_x];
            shared_N[thread_y][thread_x] = matrix_N[(phase*TILE_WIDTH + thread_y)*Width + Col];
            __syncthreads();

            for (int k = 0; k < TILE_WIDTH; ++k) {
                out_value += shared_M[thread_y][k] * shared_N[k][thread_x];
            }
            __syncthreads();
        }
        matrix_Out[Row * Width + Col] = out_value;
    }
}


void runMatrixMultiplication(
    int* matrix_M_h,
    int* matrix_N_h,
    int* matrix_Out_h,
    int Width
) {
    // Get size in bytes.
    size_t size = Width * Width * sizeof(int);

    // Load and copy matrix M and N to device memory.
    int * matrix_M_d, * matrix_N_d, * matrix_Out_d;
    hipMalloc((void***)&matrix_M_d, size);
    hipMemcpy(matrix_M_d, matrix_M_h, size, hipMemcpyHostToDevice);

    hipMalloc((void***)&matrix_N_d, size);
    hipMemcpy(matrix_N_d, matrix_N_h, size, hipMemcpyHostToDevice);

    hipMalloc((void***)&matrix_Out_d, size);

    // Invoke kernel.
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(ceil(Width / (BLOCK_SIZE * 0.1)), ceil(Width / (BLOCK_SIZE * 0.1)));
    MatrixMultiplicationKernel<<<dimGrid, dimBlock>>>(matrix_M_d, matrix_N_d, matrix_Out_d, Width);

    // Copy the output matrix from the device memory.
    hipMemcpy(matrix_Out_h, matrix_Out_d, size, hipMemcpyDeviceToHost);

    // Free device vectors.
    hipFree(matrix_M_d);
    hipFree(matrix_N_d);
    hipFree(matrix_Out_d);
}


int main() {
    // Matrices are stored in row-major order.
    int Width = 4;

    // Define identical matrices M and N where each element = 1 .. Width * Width.
    int * matrix_M = (int *) malloc(Width * Width * sizeof(int));
    int * matrix_N = (int *) malloc(Width * Width * sizeof(int));
    int * matrix_Out = (int *) malloc(Width * Width * sizeof(int));

    for (int i = 1; i <= Width * Width; ++i) {
        matrix_M[i - 1] = i;
        matrix_N[i - 1] = i;
    }

    runMatrixMultiplication(matrix_M, matrix_N, matrix_Out, Width);

    for (int i = 0; i < Width; ++i) {
        for (int j = 0; j < Width; ++j) {
            printf("%d ", matrix_Out[i * Width + j]);
        }
        printf("\n");
    }

    free(matrix_M);
    free(matrix_N);
    free(matrix_Out);

    return 0;
}